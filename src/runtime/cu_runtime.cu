#include "hip/hip_runtime.h"
/**@file cu_runtime.cu
 * @brief Implementation of low level matrix operations using CUDA kernels and
 * CuBLAS.
 */

#include <hip/hip_runtime.h>
#include <hipblas.h>
extern "C" {
    #include <cu_runtime.h>
}

#define EPSILON 1e-7

static hipblasHandle_t handle = NULL;

extern "C" nw_error_t *cu_create_context(void)
{
    hipblasStatus_t status = hipblasCreate(&handle);
    if (status != HIPBLAS_STATUS_SUCCESS)
    {
        return ERROR(ERROR_CREATE, string_create("failed to create cuda context."), NULL);
    }

    return NULL;
}

extern "C" void cu_destroy_context(void)
{
    // Automatically synchronizes the device.
    hipblasDestroy(handle);
}

extern "C" nw_error_t *cu_memory_allocate(void **pp, size_t size)
{
    CHECK_NULL_ARGUMENT(pp, "pp");

    hipError_t error = hipMallocManaged(pp, size);
    if (error != hipSuccess)
    {
        return ERROR(ERROR_MEMORY_ALLOCATION, string_create("failed to allocate %zu bytes %s.", size, hipGetErrorString(error)), NULL);
    }

    return NULL;
}

extern "C" void cu_memory_free(void *p)
{
    hipFree(p);
}

__global__ static void cu_exponential_float32(int n, const float32_t *x_data, int x_stride, float32_t *y_data, int y_stride)
{
    int i = (blockDim.x * blockIdx.x) + threadIdx.x;
    if (i < n)
    {
        y_data[i * y_stride] = expf(x_data[i * x_stride]);
    }
}

__global__ static void cu_exponential_float64(int n, const float64_t *x_data, int x_stride, float64_t *y_data, int y_stride)
{
    int i = (blockDim.x * blockIdx.x) + threadIdx.x;
    if (i < n)
    {
        y_data[i * y_stride] = exp(x_data[i * x_stride]);
    }
}

extern "C" void cu_exponential(datatype_t datatype, int64_t n, const void *x_data, int64_t x_stride, int64_t x_offset, void *y_data, int64_t y_stride, int64_t y_offset)
{
    // CUDA devs want us using ints here for minor optimization purposes, and
    // presumably because we know we're not going to overflow.
    int block_size;
    int min_grid_size;
    int grid_size;

    switch (datatype)
    {
    case FLOAT32:
        // I want to move this code into cu_context_create and store the grid
        // and block size somewhere, but presumably
        // hipOccupancyMaxPotentialBlockSize is compile time so we're only
        // losing time on the division, and darknet does something completely
        // different so I think it's best to try to understand that before doing
        // any major restructuring.
        hipOccupancyMaxPotentialBlockSize(&min_grid_size, &block_size,
                cu_exponential_float32, 0, 0);

        if (block_size == 0)
        {
            block_size = 32;
        }
        grid_size = (n + block_size - 1) / block_size;

        cu_exponential_float32<<<grid_size, block_size>>>((int) n, &((float32_t *) x_data)[x_offset], (int) x_stride, &((float32_t *) y_data)[y_offset], (int) y_stride);

        break;
    case FLOAT64:
        hipOccupancyMaxPotentialBlockSize(&min_grid_size, &block_size,
                cu_exponential_float64, 0, 0);

        if (block_size == 0)
        {
            block_size = 32;
        }
        grid_size = (n + block_size - 1) / block_size;

        cu_exponential_float64<<<grid_size, block_size>>>((int) n, &((float64_t *) x_data)[x_offset], (int) x_stride, &((float64_t *) y_data)[y_offset], (int) y_stride);

        break;
    default:
        break;
    }
}

__global__ static void cu_logarithm_float32(int n, const float32_t *x_data, int x_stride, float32_t *y_data, int y_stride)
{
    int i = (blockDim.x * blockIdx.x) + threadIdx.x;
    if (i < n)
    {
        y_data[i * y_stride] = logf(x_data[i * x_stride]);
    }
}

__global__ static void cu_logarithm_float64(int n, const float64_t *x_data, int x_stride, float64_t *y_data, int y_stride)
{
    int i = (blockDim.x * blockIdx.x) + threadIdx.x;
    if (i < n)
    {
        y_data[i * y_stride] = log(x_data[i * x_stride]);
    }
}

extern "C" void cu_logarithm(datatype_t datatype, int64_t n, const void *x_data, int64_t x_stride, int64_t x_offset, void *y_data, int64_t y_stride, int64_t y_offset)
{
    int block_size;
    int min_grid_size;
    int grid_size;

    switch (datatype)
    {
    case FLOAT32:
        hipOccupancyMaxPotentialBlockSize(&min_grid_size, &block_size,
                cu_logarithm_float32, 0, 0);

        if (block_size == 0)
        {
            block_size = 32;
        }
        grid_size = (n + block_size - 1) / block_size;

        cu_logarithm_float32<<<grid_size, block_size>>>((int) n, &((float32_t *) x_data)[x_offset], (int) x_stride, &((float32_t *) y_data)[y_offset], (int) y_stride);
        break;

    case FLOAT64:
        hipOccupancyMaxPotentialBlockSize(&min_grid_size, &block_size,
                cu_logarithm_float64, 0, 0);

        if (block_size == 0)
        {
            block_size = 32;
        }
        grid_size = (n + block_size - 1) / block_size;

        cu_logarithm_float64<<<grid_size, block_size>>>((int) n, &((float64_t *) x_data)[x_offset], (int) x_stride, &((float64_t *) y_data)[y_offset], (int) y_stride);

        break;
    default:
        break;
    }
}

__global__ static void cu_sine_float32(int n, const float32_t *x_data, int x_stride, float32_t *y_data, int y_stride)
{
    int i = (blockDim.x * blockIdx.x) + threadIdx.x;
    if (i < n)
    {
        y_data[i * y_stride] = sinf(x_data[i * x_stride]);
    }
}

__global__ static void cu_sine_float64(int n, const float64_t *x_data, int x_stride, float64_t *y_data, int y_stride)
{
    int i = (blockDim.x * blockIdx.x) + threadIdx.x;
    if (i < n)
    {
        y_data[i * y_stride] = sin(x_data[i * x_stride]);
    }
}

extern "C" void cu_sine(datatype_t datatype, int64_t n, const void *x_data, int64_t x_stride, int64_t x_offset, void *y_data, int64_t y_stride, int64_t y_offset)
{
    int block_size;
    int min_grid_size;
    int grid_size;

    switch (datatype)
    {
    case FLOAT32:
        hipOccupancyMaxPotentialBlockSize(&min_grid_size, &block_size,
                cu_sine_float32, 0, 0);

        if (block_size == 0)
        {
            block_size = 32;
        }
        grid_size = (n + block_size - 1) / block_size;

        cu_sine_float32<<<grid_size, block_size>>>((int) n, &((float32_t *) x_data)[x_offset], (int) x_stride, &((float32_t *) y_data)[y_offset], (int) y_stride);

        break;
    case FLOAT64:
        hipOccupancyMaxPotentialBlockSize(&min_grid_size, &block_size,
                cu_sine_float64, 0, 0);

        if (block_size == 0)
        {
            block_size = 32;
        }
        grid_size = (n + block_size - 1) / block_size;

        cu_sine_float64<<<grid_size, block_size>>>((int) n, &((float64_t *) x_data)[x_offset], (int) x_stride, &((float64_t *) y_data)[y_offset], (int) y_stride);

        break;
    default:
        break;
    }
}

__global__ static void cu_cosine_float32(int n, const float32_t *x_data, int x_stride, float32_t *y_data, int y_stride)
{
    int i = (blockDim.x * blockIdx.x) + threadIdx.x;
    if (i < n)
    {
        y_data[i * y_stride] = cosf(x_data[i * x_stride]);
    }
}

__global__ static void cu_cosine_float64(int n, const float64_t *x_data, int x_stride, float64_t *y_data, int y_stride)
{
    int i = (blockDim.x * blockIdx.x) + threadIdx.x;
    if (i < n)
    {
        y_data[i * y_stride] = cos(x_data[i * x_stride]);
    }
}

extern "C" void cu_cosine(datatype_t datatype, int64_t n, const void *x_data, int64_t x_stride, int64_t x_offset, void *y_data, int64_t y_stride, int64_t y_offset)
{
    int block_size;
    int min_grid_size;
    int grid_size;

    switch (datatype)
    {
    case FLOAT32:
        hipOccupancyMaxPotentialBlockSize(&min_grid_size, &block_size,
                cu_cosine_float32, 0, 0);

        if (block_size == 0)
        {
            block_size = 32;
        }
        grid_size = (n + block_size - 1) / block_size;

        cu_cosine_float32<<<grid_size, block_size>>>((int) n, &((float32_t *) x_data)[x_offset], (int) x_stride, &((float32_t *) y_data)[y_offset], (int) y_stride);

        break;
    case FLOAT64:
        hipOccupancyMaxPotentialBlockSize(&min_grid_size, &block_size,
                cu_cosine_float64, 0, 0);

        if (block_size == 0)
        {
            block_size = 32;
        }
        cu_cosine_float64<<<grid_size, block_size>>>((int) n, &((float64_t *) x_data)[x_offset], (int) x_stride, &((float64_t *) y_data)[y_offset], (int) y_stride);

        break;
    default:
        break;
    }
}

__global__ static void cu_square_root_float32(int n, const float32_t *x_data, int x_stride, float32_t *y_data, int y_stride)
{
    int i = (blockDim.x * blockIdx.x) + threadIdx.x;
    if (i < n)
    {
        y_data[i * y_stride] = sqrtf(x_data[i * x_stride]);
    }
}

__global__ static void cu_square_root_float64(int n, const float64_t *x_data, int x_stride, float64_t *y_data, int y_stride)
{
    int i = (blockDim.x * blockIdx.x) + threadIdx.x;
    if (i < n)
    {
        y_data[i * y_stride] = sqrt(x_data[i * x_stride]);
    }
}

extern "C" void cu_square_root(datatype_t datatype, int64_t n, const void *x_data, int64_t x_stride, int64_t x_offset, void *y_data, int64_t y_stride, int64_t y_offset)
{
    int block_size;
    int min_grid_size;
    int grid_size;

    switch (datatype)
    {
    case FLOAT32:
        hipOccupancyMaxPotentialBlockSize(&min_grid_size, &block_size,
                cu_square_root_float32, 0, 0);

        if (block_size == 0)
        {
            block_size = 32;
        }
        grid_size = (n + block_size - 1) / block_size;

        cu_square_root_float32<<<grid_size, block_size>>>((int) n, &((float32_t *) x_data)[x_offset], (int) x_stride, &((float32_t *) y_data)[y_offset], (int) y_stride);

        break;
    case FLOAT64:
        hipOccupancyMaxPotentialBlockSize(&min_grid_size, &block_size,
                cu_square_root_float64, 0, 0);

        if (block_size == 0)
        {
            block_size = 32;
        }
        grid_size = (n + block_size - 1) / block_size;

        cu_square_root_float64<<<grid_size, block_size>>>((int) n, &((float64_t *) x_data)[x_offset], (int) x_stride, &((float64_t *) y_data)[y_offset], (int) y_stride);

        break;
    default:
        break;
    }
}

__global__ static void cu_reciprocal_float32(int n, const float32_t *x_data, int x_stride, float32_t *y_data, int y_stride)
{
    int i = (blockDim.x * blockIdx.x) + threadIdx.x;
    if (i < n)
    {
        y_data[i * y_stride] = 1. / x_data[i * x_stride];
    }
}

__global__ static void cu_reciprocal_float64(int n, const float64_t *x_data, int x_stride, float64_t *y_data, int y_stride)
{
    int i = (blockDim.x * blockIdx.x) + threadIdx.x;
    if (i < n)
    {
        y_data[i * y_stride] = 1. / x_data[i * x_stride];
    }
}

extern "C" void cu_reciprocal(datatype_t datatype, int64_t n, const void *x_data, int64_t x_stride, int64_t x_offset, void *y_data, int64_t y_stride, int64_t y_offset)
{
    int block_size;
    int min_grid_size;
    int grid_size;

    switch (datatype)
    {
    case FLOAT32:
        hipOccupancyMaxPotentialBlockSize(&min_grid_size, &block_size,
                cu_reciprocal_float32, 0, 0);

        if (block_size == 0)
        {
            block_size = 32;
        }
        grid_size = (n + block_size - 1) / block_size;

        cu_reciprocal_float32<<<grid_size, block_size>>>((int) n, &((float32_t *) x_data)[x_offset], (int) x_stride, &((float32_t *) y_data)[y_offset], (int) y_stride);

        break;
    case FLOAT64:
        hipOccupancyMaxPotentialBlockSize(&min_grid_size, &block_size,
                cu_reciprocal_float64, 0, 0);

        if (block_size == 0)
        {
            block_size = 32;
        }
        grid_size = (n + block_size - 1) / block_size;

        cu_reciprocal_float64<<<grid_size, block_size>>>((int) n, &((float64_t *) x_data)[x_offset], (int) x_stride, &((float64_t *) y_data)[y_offset], (int) y_stride);

        break;
    default:
        break;
    }
}

extern "C" void cu_copy(datatype_t datatype, int64_t n, const void *x_data, int64_t x_stride, int64_t x_offset, void *y_data, int64_t y_stride, int64_t y_offset)
{
    switch (datatype)
    {
    case FLOAT32:
        hipblasScopy(handle, (int) n, &((float32_t *) x_data)[x_offset], (int) x_stride, &((float32_t *) y_data)[y_offset], (int) y_stride);
        hipDeviceSynchronize();
        break;
    case FLOAT64:
        hipblasDcopy(handle, (int) n, &((float64_t *) x_data)[x_offset], (int) x_stride, &((float64_t *) y_data)[y_offset], (int) y_stride);
        hipDeviceSynchronize();
        break;
    default:
        break;
    }
}

__global__ static void cu_negation_float32(int n, const float32_t *x_data, int x_stride, float32_t *y_data, int y_stride)
{
    int i = (blockDim.x * blockIdx.x) + threadIdx.x;
    if (i < n)
    {
        y_data[i * y_stride] = -x_data[i * x_stride];
    }
}

__global__ static void cu_negation_float64(int n, const float64_t *x_data, int x_stride, float64_t *y_data, int y_stride)
{
    int i = (blockDim.x * blockIdx.x) + threadIdx.x;
    if (i < n)
    {
        y_data[i * y_stride] = -x_data[i * x_stride];
    }
}

void cu_negation(datatype_t datatype,
                 int64_t n,
                 const void *x_data,
                 int64_t x_stride,
                 int64_t x_offset,
                 void *y_data,
                 int64_t y_stride,
                 int64_t y_offset)
{
    int block_size;
    int min_grid_size;
    int grid_size;

    switch (datatype)
    {
    case FLOAT32:
        hipOccupancyMaxPotentialBlockSize(&min_grid_size, &block_size,
                cu_negation_float32, 0, 0);

        if (block_size == 0)
        {
            block_size = 32;
        }
        grid_size = (n + block_size - 1) / block_size;

        cu_negation_float32<<<grid_size, block_size>>>((int) n, &((float32_t *) x_data)[x_offset], (int) x_stride, &((float32_t *) y_data)[y_offset], (int) y_stride);

        break;
    case FLOAT64:
        hipOccupancyMaxPotentialBlockSize(&min_grid_size, &block_size,
                cu_negation_float64, 0, 0);

        if (block_size == 0)
        {
            block_size = 32;
        }
        grid_size = (n + block_size - 1) / block_size;

        cu_negation_float64<<<grid_size, block_size>>>((int) n, &((float64_t *) x_data)[x_offset], (int) x_stride, &((float64_t *) y_data)[y_offset], (int) y_stride);

        break;
    default:
        break;
    }
}

__global__ static void cu_rectified_linear_float32(int n, const float32_t *x_data, int x_stride, float32_t *y_data, int y_stride)
{
    int i = (blockDim.x * blockIdx.x) + threadIdx.x;
    if (i < n)
    {
        float32_t value = x_data[i * x_stride];
        y_data[i * y_stride] = (value > 0.0) ? value : (float32_t) 0.0;
    }
}

__global__ static void cu_rectified_linear_float64(int n, const float64_t *x_data, int x_stride, float64_t *y_data, int y_stride)
{
    int i = (blockDim.x * blockIdx.x) + threadIdx.x;
    if (i < n)
    {
        float64_t value = x_data[i * x_stride];
        y_data[i * y_stride] = (value > 0.0) ? value : (float64_t) 0.0;
    }
}

extern "C" void cu_rectified_linear(datatype_t datatype, int64_t n, const void *x_data, int64_t x_stride, int64_t x_offset, void *y_data, int64_t y_stride, int64_t y_offset)
{
    int block_size;
    int min_grid_size;
    int grid_size;

    switch (datatype)
    {
    case FLOAT32:
        hipOccupancyMaxPotentialBlockSize(&min_grid_size, &block_size,
                cu_rectified_linear_float32, 0, 0);

        if (block_size == 0)
        {
            block_size = 32;
        }
        grid_size = (n + block_size - 1) / block_size;

        cu_rectified_linear_float32<<<grid_size, block_size>>>((int) n, &((float32_t *) x_data)[x_offset], (int) x_stride, &((float32_t *) y_data)[y_offset], (int) y_stride);

        break;
    case FLOAT64:
        hipOccupancyMaxPotentialBlockSize(&min_grid_size, &block_size,
                cu_rectified_linear_float64, 0, 0);

        if (block_size == 0)
        {
            block_size = 32;
        }
        grid_size = (n + block_size - 1) / block_size;

        cu_rectified_linear_float64<<<grid_size, block_size>>>((int) n, &((float64_t *) x_data)[x_offset], (int) x_stride, &((float64_t *) y_data)[y_offset], (int) y_stride);

        break;
    default:
        break;
    }
}

__global__ static void cu_sigmoid_float32(int n, const float32_t *x_data, int x_stride, float32_t *y_data, int y_stride)
{
    int i = (blockDim.x * blockIdx.x) + threadIdx.x;
    if (i < n)
    {
        float32_t x = x_data[i * x_stride];
        if (x >= 0)
        {
            y_data[i * y_stride] = (float32_t) 1.0 / ((float32_t) 1.0 + expf(-x)); 
        }
        else
        {
            y_data[i * y_stride] = expf(x) / ((float32_t) 1.0 + expf(x)); 
        }
    }
}

__global__ static void cu_sigmoid_float64(int n, const float64_t *x_data, int x_stride, float64_t *y_data, int y_stride)
{
    int i = (blockDim.x * blockIdx.x) + threadIdx.x;
    if (i < n)
    {
        float64_t x = x_data[i * x_stride];
        if (x >= 0)
        {
            y_data[i * y_stride] = (float64_t) 1.0 / ((float64_t) 1.0 + exp(-x)); 
        }
        else
        {
            y_data[i * y_stride] = exp(x) / ((float64_t) 1.0 + exp(x)); 
        }
    }
}

extern "C" void cu_sigmoid(datatype_t datatype, int64_t n, const void *x_data, int64_t x_stride, int64_t x_offset, void *y_data, int64_t y_stride, int64_t y_offset)
{
    int block_size;
    int min_grid_size;
    int grid_size;

    switch (datatype)
    {
    case FLOAT32:
        hipOccupancyMaxPotentialBlockSize(&min_grid_size, &block_size,
                cu_sigmoid_float32, 0, 0);

        if (block_size == 0)
        {
            block_size = 32;
        }
        grid_size = (n + block_size - 1) / block_size;

        cu_sigmoid_float32<<<grid_size, block_size>>>((int) n, &((float32_t *) x_data)[x_offset], (int) x_stride, &((float32_t *) y_data)[y_offset], (int) y_stride);

        break;
    case FLOAT64:
        hipOccupancyMaxPotentialBlockSize(&min_grid_size, &block_size,
                cu_sigmoid_float64, 0, 0);

        if (block_size == 0)
        {
            block_size = 32;
        }
        grid_size = (n + block_size - 1) / block_size;

        cu_sigmoid_float64<<<grid_size, block_size>>>((int) n, &((float64_t *) x_data)[x_offset], (int) x_stride, &((float64_t *) y_data)[y_offset], (int) y_stride);

        break;
    default:
        break;
    }
}

extern "C" static void cu_addition_float32(int n,
                                           const float32_t *x_data,
                                           int x_stride,
                                           const float32_t *y_data,
                                           int y_stride,
                                           float32_t *z_data,
                                           int z_stride)
{
    float32_t *alpha;
    hipMallocManaged((void **) &alpha, sizeof(float32_t));
    *alpha = (float32_t) 1.0;
    hipblasScopy(handle, n, x_data, x_stride, z_data, z_stride);
    hipDeviceSynchronize();
    hipblasSaxpy(handle, n, alpha, y_data, y_stride, z_data, z_stride);
    hipDeviceSynchronize();
    hipFree(alpha);

}

extern "C" static void cu_addition_float64(int n,
                                           const float64_t *x_data,
                                           int x_stride,
                                           const float64_t *y_data,
                                           int y_stride,
                                           double *z_data,
                                           float64_t z_stride)
{
    float64_t *alpha;
    hipMallocManaged((void **) &alpha, sizeof(float64_t));
    *alpha = (float64_t) 1.0;
    hipblasDcopy(handle, n, x_data, x_stride, z_data, z_stride);
    hipDeviceSynchronize();
    hipblasDaxpy(handle, n, alpha, y_data, y_stride, z_data, z_stride);
    hipDeviceSynchronize();
    hipFree(alpha);

}

extern "C" void cu_addition(datatype_t datatype,
                            int64_t n,
                            const void *x_data,
                            int64_t x_stride,
                            int64_t x_offset,
                            const void *y_data,
                            int64_t y_stride,
                            int64_t y_offset,
                            void *z_data,
                            int64_t z_stride,
                            int64_t z_offset)
{
    switch (datatype)
    {
    case FLOAT32:
        cu_addition_float32((int) n, 
                            &((float32_t *) x_data)[x_offset], 
                            (int) x_stride,
                            &((float32_t *) y_data)[y_offset],
                            (int) y_stride,
                            &((float32_t *) z_data)[z_offset],
                            (int) z_stride);
        break;
    case FLOAT64:
        cu_addition_float64((int) n, 
                            &((float64_t *) x_data)[x_offset], 
                            (int) x_stride,
                            &((float64_t *) y_data)[y_offset],
                            (int) y_stride,
                            &((float64_t *) z_data)[z_offset],
                            (int) z_stride);
        break;
    default:
        break;
    }
}

extern "C" static void cu_subtraction_float32(int n,
                                              const float32_t *x_data,
                                              int x_stride,
                                              const float32_t *y_data,
                                              int y_stride,
                                              float32_t *z_data,
                                              int z_stride)
{
    float32_t *alpha;
    hipMallocManaged((void **) &alpha, sizeof(float32_t));
    *alpha = (float32_t) -1.0;
    hipblasScopy(handle, n, x_data, x_stride, z_data, z_stride);
    hipDeviceSynchronize();
    hipblasSaxpy(handle, n, alpha, y_data, y_stride, z_data, z_stride);
    hipDeviceSynchronize();
    hipFree(alpha);
}

extern "C" static void cu_subtraction_float64(int n,
                                              const float64_t *x_data,
                                              int x_stride,
                                              const float64_t *y_data,
                                              int y_stride,
                                              float64_t *z_data,
                                              int z_stride)
{
    float64_t *alpha;
    hipMallocManaged((void **) &alpha, sizeof(float64_t));
    *alpha = (float64_t) -1.0;
    hipblasDcopy(handle, n, x_data, x_stride, z_data, z_stride);
    hipDeviceSynchronize();
    hipblasDaxpy(handle, n, alpha, y_data, y_stride, z_data, z_stride);
    hipDeviceSynchronize();
    hipFree(alpha);

}

extern "C" void cu_subtraction(datatype_t datatype,
                               int64_t n,
                               const void *x_data,
                               int64_t x_stride,
                               int64_t x_offset,
                               const void *y_data,
                               int64_t y_stride,
                               int64_t y_offset,
                               void *z_data,
                               int64_t z_stride,
                               int64_t z_offset)
{
    switch (datatype)
    {
    case FLOAT32:
        cu_subtraction_float32((int) n, 
                               &((float32_t *) x_data)[x_offset], 
                               (int) x_stride,
                               &((float32_t *) y_data)[y_offset],
                               (int) y_stride,
                               &((float32_t *) z_data)[z_offset],
                               (int) z_stride);
        break;
    case FLOAT64:
        cu_subtraction_float64((int) n, 
                               &((float64_t *) x_data)[x_offset], 
                               (int) x_stride,
                               &((float64_t *) y_data)[y_offset],
                               (int) y_stride,
                               &((float64_t *) z_data)[z_offset],
                               (int) z_stride);
        break;
    default:
        break;
    }
}

__global__ static void cu_multiplication_float32(int n,
                                                 const float32_t *x_data,
                                                 int x_stride,
                                                 const float32_t *y_data,
                                                 int y_stride,
                                                 float32_t *z_data,
                                                 int z_stride)
{
    int i = (blockDim.x * blockIdx.x) + threadIdx.x;
    if (i < n)
    {
        z_data[i * z_stride] = x_data[i * x_stride] * y_data[i * y_stride];
    }
}

__global__ static void cu_multiplication_float64(int n,
                                                 const float64_t *x_data,
                                                 int x_stride,
                                                 const float64_t *y_data,
                                                 int y_stride,
                                                 float64_t *z_data,
                                                 int z_stride)
{
    int i = (blockDim.x * blockIdx.x) + threadIdx.x;
    if (i < n)
    {
        z_data[i * z_stride] = x_data[i * x_stride] * y_data[i * y_stride];
    }
}

extern "C" void cu_multiplication(datatype_t datatype,
                                  int64_t n,
                                  const void *x_data,
                                  int64_t x_stride,
                                  int64_t x_offset,
                                  const void *y_data,
                                  int64_t y_stride,
                                  int64_t y_offset,
                                  void *z_data,
                                  int64_t z_stride,
                                  int64_t z_offset)
{
    int block_size;
    int min_grid_size;
    int grid_size;

    switch (datatype)
    {
    case FLOAT32:
        hipOccupancyMaxPotentialBlockSize(&min_grid_size, &block_size,
                cu_multiplication_float32, 0, 0);

        if (block_size == 0)
        {
            block_size = 32;
        }
        grid_size = (n + block_size - 1) / block_size;

        cu_multiplication_float32<<<grid_size, block_size>>>((int) n,
                                  &((float32_t *) x_data)[x_offset],
                                  (int) x_stride,
                                  &((float32_t *) y_data)[y_offset],
                                  (int) y_stride,
                                  &((float32_t *) z_data)[z_offset],
                                  (int) z_stride);

        break;
    case FLOAT64:
        hipOccupancyMaxPotentialBlockSize(&min_grid_size, &block_size,
                cu_multiplication_float64, 0, 0);

        if (block_size == 0)
        {
            block_size = 32;
        }
        grid_size = (n + block_size - 1) / block_size;

        cu_multiplication_float64<<<grid_size, block_size>>>((int) n,
                                  &((float64_t *) x_data)[x_offset],
                                  (int) x_stride,
                                  &((float64_t *) y_data)[y_offset],
                                  (int) y_stride,
                                  &((float64_t *) z_data)[z_offset],
                                  (int) z_stride);

        break;
    default:
        break;
    }
}

__global__ static void cu_division_float32(int n, const float32_t *x_data, int x_stride, const float32_t *y_data, int y_stride, float32_t *z_data, int z_stride)
{
    int i = (blockDim.x * blockIdx.x) + threadIdx.x;
    if (i < n)
    {
        z_data[i * z_stride] = x_data[i * x_stride] / y_data[i * y_stride];
    }
}

__global__ static void cu_division_float64(int n, const float64_t *x_data, int x_stride, const float64_t *y_data, int y_stride, float64_t *z_data, int z_stride)
{
    int i = (blockDim.x * blockIdx.x) + threadIdx.x;
    if (i < n)
    {
        z_data[i * z_stride] = x_data[i * x_stride] / y_data[i * y_stride];
    }
}

extern "C" void cu_division(datatype_t datatype, int64_t n, const void *x_data, int64_t x_stride, int64_t x_offset, const void *y_data, int64_t y_stride, int64_t y_offset, void *z_data, int64_t z_stride, int64_t z_offset)
{
    int block_size;
    int min_grid_size;
    int grid_size;

    switch (datatype)
    {
    case FLOAT32:
        hipOccupancyMaxPotentialBlockSize(&min_grid_size, &block_size,
                cu_division_float32, 0, 0);

        if (block_size == 0)
        {
            block_size = 32;
        }
        grid_size = (n + block_size - 1) / block_size;

        cu_division_float32<<<grid_size, block_size>>>((int) n, &((float32_t *) x_data)[x_offset], (int) x_stride, &((float32_t *) y_data)[y_offset], (int) y_stride, &((float32_t *) z_data)[z_offset], (int) z_stride);

        break;
    case FLOAT64:
        hipOccupancyMaxPotentialBlockSize(&min_grid_size, &block_size,
                cu_division_float64, 0, 0);

        if (block_size == 0)
        {
            block_size = 32;
        }
        grid_size = (n + block_size - 1) / block_size;

        cu_division_float64<<<grid_size, block_size>>>((int) n, &((float64_t *) x_data)[x_offset], (int) x_stride, &((float64_t *) y_data)[y_offset], (int) y_stride, &((float64_t *) z_data)[z_offset], (int) z_stride);

        break;
    default:
        break;
    }
}

__global__ static void cu_power_float32(int n, const float32_t *x_data, int x_stride, const float32_t *y_data, int y_stride, float32_t *z_data, int z_stride)
{
    int i = (blockDim.x * blockIdx.x) + threadIdx.x;
    if (i < n)
    {
        z_data[i * z_stride] = powf(x_data[i * x_stride], y_data[i * y_stride]);
    }
}

__global__ static void cu_power_float64(int n, const float64_t *x_data, int x_stride, const float64_t *y_data, int y_stride, float64_t *z_data, int z_stride)
{
    int i = (blockDim.x * blockIdx.x) + threadIdx.x;
    if (i < n)
    {
        z_data[i * z_stride] = pow(x_data[i * x_stride], y_data[i * y_stride]);
    }
}

extern "C" void cu_power(datatype_t datatype, int64_t n, const void *x_data, int64_t x_stride, int64_t x_offset, const void *y_data, int64_t y_stride, int64_t y_offset, void *z_data, int64_t z_stride, int64_t z_offset)
{
    int block_size;
    int min_grid_size;
    int grid_size;

    switch (datatype)
    {
    case FLOAT32:
        hipOccupancyMaxPotentialBlockSize(&min_grid_size, &block_size,
                cu_power_float32, 0, 0);

        if (block_size == 0)
        {
            block_size = 32;
        }
        grid_size = (n + block_size - 1) / block_size;

        cu_power_float32<<<grid_size, block_size>>>((int) n, &((float32_t *) x_data)[x_offset], (int) x_stride, &((float32_t *) y_data)[y_offset], (int) y_stride, &((float32_t *) z_data)[z_offset], (int) z_stride);

        break;
    case FLOAT64:
        hipOccupancyMaxPotentialBlockSize(&min_grid_size, &block_size,
                cu_power_float64, 0, 0);

        if (block_size == 0)
        {
            block_size = 32;
        }
        grid_size = (n + block_size - 1) / block_size;

        cu_power_float64<<<grid_size, block_size>>>((int) n, &((float64_t *) x_data)[x_offset], (int) x_stride, &((float64_t *) y_data)[y_offset], (int) y_stride, &((float64_t *) z_data)[z_offset], (int) z_stride);

        break;
    default:
        break;
    }
}

__global__ static void cu_compare_equal_float32(int n, const float32_t *x_data, int x_stride, const float32_t *y_data, int y_stride, float32_t *z_data, int z_stride)
{
    float32_t x, y;
    int i = (blockDim.x * blockIdx.x) + threadIdx.x;
    if (i < n)
    {
        x = x_data[i * x_stride];
        y = y_data[i * y_stride];
        z_data[i * z_stride] = fabsf(x - y) < EPSILON ? (float32_t) 1.0 : (float32_t) 0.0;
    }
}

__global__ static void cu_compare_equal_float64(int n, const float64_t *x_data, int x_stride, const float64_t *y_data, int y_stride, float64_t *z_data, int z_stride)
{
    float64_t x, y;
    int i = (blockDim.x * blockIdx.x) + threadIdx.x;
    if (i < n)
    {
        x = x_data[i * x_stride];
        y = y_data[i * y_stride];
        z_data[i * z_stride] = fabs(x - y) < EPSILON ? (float64_t) 1.0 : (float64_t) 0.0;
    }
}

extern "C" void cu_compare_equal(datatype_t datatype, int64_t n, const void *x_data, int64_t x_stride, int64_t x_offset, const void *y_data, int64_t y_stride, int64_t y_offset, void *z_data, int64_t z_stride, int64_t z_offset)
{
    int block_size;
    int min_grid_size;
    int grid_size;

    switch (datatype)
    {
    case FLOAT32:
        hipOccupancyMaxPotentialBlockSize(&min_grid_size, &block_size,
                cu_compare_equal_float32, 0, 0);

        if (block_size == 0)
        {
            block_size = 32;
        }
        grid_size = (n + block_size - 1) / block_size;

        cu_compare_equal_float32<<<grid_size, block_size>>>((int) n, &((float32_t *) x_data)[x_offset], (int) x_stride, &((float32_t *) y_data)[y_offset], (int) y_stride, &((float32_t *) z_data)[z_offset], (int) z_stride);

        break;
    case FLOAT64:
        hipOccupancyMaxPotentialBlockSize(&min_grid_size, &block_size,
                cu_compare_equal_float64, 0, 0);

        if (block_size == 0)
        {
            block_size = 32;
        }
        grid_size = (n + block_size - 1) / block_size;

        cu_compare_equal_float64<<<grid_size, block_size>>>((int) n, &((float64_t *) x_data)[x_offset], (int) x_stride, &((float64_t *) y_data)[y_offset], (int) y_stride, &((float64_t *) z_data)[z_offset], (int) z_stride);

        break;
    default:
        break;
    }
}

__global__ static void cu_compare_greater_float32(int n, const float32_t *x_data, int x_stride, const float32_t *y_data, int y_stride, float32_t *z_data, int z_stride)
{
    int i = (blockDim.x * blockIdx.x) + threadIdx.x;
    if (i < n)
    {
        z_data[i * z_stride] = (x_data[i * x_stride] > y_data[i * y_stride]) ? (float32_t) 1.0 : (float32_t) 0.0;
    }
}

__global__ static void cu_compare_greater_float64(int n, const float64_t *x_data, int x_stride, const float64_t *y_data, int y_stride, float64_t *z_data, int z_stride)
{
    int i = (blockDim.x * blockIdx.x) + threadIdx.x;
    if (i < n)
    {
        z_data[i * z_stride] = (x_data[i * x_stride] > y_data[i * y_stride]) ? (float64_t) 1.0 : (float64_t) 0.0;
    }
}

extern "C" void cu_compare_greater(datatype_t datatype, int64_t n, const void *x_data, int64_t x_stride, int64_t x_offset, const void *y_data, int64_t y_stride, int64_t y_offset, void *z_data, int64_t z_stride, int64_t z_offset)
{
    int block_size;
    int min_grid_size;
    int grid_size;

    switch (datatype)
    {
    case FLOAT32:
        hipOccupancyMaxPotentialBlockSize(&min_grid_size, &block_size,
                cu_compare_greater_float32, 0, 0);

        if (block_size == 0)
        {
            block_size = 32;
        }
        grid_size = (n + block_size - 1) / block_size;

        cu_compare_greater_float32<<<grid_size, block_size>>>((int) n, &((float32_t *) x_data)[x_offset], (int) x_stride, &((float32_t *) y_data)[y_offset], (int) y_stride, &((float32_t *) z_data)[z_offset], (int) z_stride);

        break;
    case FLOAT64:
        hipOccupancyMaxPotentialBlockSize(&min_grid_size, &block_size,
                cu_compare_greater_float64, 0, 0);

        if (block_size == 0)
        {
            block_size = 32;
        }
        grid_size = (n + block_size - 1) / block_size;

        cu_compare_greater_float64<<<grid_size, block_size>>>((int) n, &((float64_t *) x_data)[x_offset], (int) x_stride, &((float64_t *) y_data)[y_offset], (int) y_stride, &((float64_t *) z_data)[z_offset], (int) z_stride);

        break;
    default:
        break;
    }
}

extern "C" void cu_matrix_multiplication_float32(datatype_t datatype,
                                                 int64_t m,
                                                 int64_t k,
                                                 int64_t n,
                                                 bool_t x_transpose,
                                                 bool_t y_transpose,
                                                 const float32_t *x_data,
                                                 const float32_t *y_data,
                                                 float32_t *z_data)
{
    float32_t *alpha;
    float32_t *beta;
    hipMallocManaged((void **) &alpha, sizeof(float32_t));
    hipMallocManaged((void **) &beta, sizeof(float32_t));
    *alpha = (float32_t) 1.0;
    *beta = (float32_t) 0.0;
    hipblasSgemm(handle,
                   y_transpose ? HIPBLAS_OP_T : HIPBLAS_OP_N,
                   x_transpose ? HIPBLAS_OP_T : HIPBLAS_OP_N,
                   n, m, k, alpha,
                   y_data, n, x_data, 
                   k, beta, z_data, n);
    hipDeviceSynchronize();
    hipFree(alpha);
    hipFree(beta);
}

extern "C" void cu_matrix_multiplication_float64(datatype_t datatype,
                                                 int64_t m,
                                                 int64_t k,
                                                 int64_t n,
                                                 bool_t x_transpose,
                                                 bool_t y_transpose,
                                                 const float64_t *x_data,
                                                 const float64_t *y_data,
                                                 float64_t *z_data)
{
    float64_t *alpha;
    float64_t *beta;
    hipMallocManaged((void **) &alpha, sizeof(float64_t));
    hipMallocManaged((void **) &beta, sizeof(float64_t));
    *alpha = (float64_t) 1.0;
    *beta = (float64_t) 0.0;
    hipblasDgemm(handle,
                   y_transpose ? HIPBLAS_OP_T : HIPBLAS_OP_N,
                   x_transpose ? HIPBLAS_OP_T : HIPBLAS_OP_N,
                   n, m, k, alpha,
                   y_data, n, x_data, 
                   k, beta, z_data, n);
    hipDeviceSynchronize();
    hipFree(alpha);
    hipFree(beta);
}

extern "C" void cu_matrix_multiplication(datatype_t datatype,
                                         int64_t m,
                                         int64_t k,
                                         int64_t n,
                                         bool_t x_transpose,
                                         bool_t y_transpose,
                                         const void *x_data,
                                         int64_t x_offset,
                                         const void *y_data,
                                         int64_t y_offset,
                                         void *z_data,
                                         int64_t z_offset)
{
    switch (datatype)
    {
    case FLOAT32:
        cu_matrix_multiplication_float32(datatype,
                                         m,
                                         k,
                                         n,
                                         x_transpose,
                                         y_transpose,
                                         &((float32_t *) x_data)[x_offset],
                                         &((float32_t *) y_data)[y_offset],
                                         &((float32_t *) z_data)[z_offset]);
        break;
    case FLOAT64:
        cu_matrix_multiplication_float64(datatype,
                                         m,
                                         k,
                                         n,
                                         x_transpose,
                                         y_transpose,
                                         &((float64_t *) x_data)[x_offset],
                                         &((float64_t *) y_data)[y_offset],
                                         &((float64_t *) z_data)[z_offset]);
        break;
    default:
        break;
    }
}

extern "C" static void cu_summation_float32(int n, const float32_t *x_data, int x_stride, float32_t *y_data)
{
    float32_t *temp;
    hipMallocManaged((void **) &temp, sizeof(float32_t));
    *temp = (float32_t) 1.0;
    hipblasSdot(handle, n, x_data, x_stride, temp, (int) 0, y_data);
    hipDeviceSynchronize();
    hipFree(temp);
}

extern "C" static void cu_summation_float64(int n, const float64_t *x_data, int x_stride, float64_t *y_data)
{
    float64_t *temp;
    hipMallocManaged((void **) &temp, sizeof(float64_t));
    *temp = (float64_t) 1.0;
    hipblasDdot(handle, n, x_data, x_stride, temp, (int) 0, y_data);
    hipDeviceSynchronize();
    hipFree(temp);
}

extern "C" void cu_summation(datatype_t datatype, int64_t n, const void *x_data, int64_t x_stride, int64_t x_offset, void *y_data, int64_t y_offset)
{
    switch (datatype)
    {
    case FLOAT32:
        cu_summation_float32((int) n, &((float32_t *) x_data)[x_offset], (int) x_stride, &((float32_t *) y_data)[y_offset]);
        break;
    case FLOAT64:
        cu_summation_float64((int) n, &((float64_t *) x_data)[x_offset], (int) x_stride, &((float64_t *) y_data)[y_offset]);
        break;
    default:
        break;
    }
}

__global__ static void cu_maximum_float32(int n, const float32_t *x_data, int x_stride, float32_t *y_data)
{
    float32_t maximum = *x_data;
    int i = (blockDim.x * blockIdx.x) + threadIdx.x + 1;
    if (i < n)
    {
        float32_t candidate = x_data[i * x_stride];
        if (maximum < candidate)
        {
            maximum = candidate;
        }
    }
    *y_data = maximum;
}

__global__ static void cu_maximum_float64(int n, const float64_t *x_data, int x_stride, float64_t *y_data)
{
    float64_t maximum = *x_data;
    int i = (blockDim.x * blockIdx.x) + threadIdx.x + 1;
    if (i < n)
    {
        float64_t candidate = x_data[i * x_stride];
        if (maximum < candidate)
        {
            maximum = candidate;
        }
    }
    *y_data = maximum;
}

extern "C" void cu_maximum(datatype_t datatype, int64_t n, const void *x_data, int64_t x_stride, int64_t x_offset, void *y_data, int64_t y_offset)
{
    int block_size;
    int min_grid_size;
    int grid_size;

    switch (datatype)
    {
    case FLOAT32:
        hipOccupancyMaxPotentialBlockSize(&min_grid_size, &block_size,
                cu_maximum_float32, 0, 0);

        if (block_size == 0)
        {
            block_size = 32;
        }
        grid_size = (n + block_size - 1) / block_size;

        cu_maximum_float32<<<grid_size, block_size>>>((int) n, &((float32_t *) x_data)[x_offset], (int) x_stride, &((float32_t *) y_data)[y_offset]);

        break;
    case FLOAT64:
        hipOccupancyMaxPotentialBlockSize(&min_grid_size, &block_size,
                cu_maximum_float64, 0, 0);

        if (block_size == 0)
        {
            block_size = 32;
        }
        grid_size = (n + block_size - 1) / block_size;

        cu_maximum_float64<<<grid_size, block_size>>>((int) n, &((float64_t *) x_data)[x_offset], (int) x_stride, &((float64_t *) y_data)[y_offset]);

        break;
    default:
        break;
    }
}
