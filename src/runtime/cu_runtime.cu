#include <hip/hip_runtime.h>
#include <hipblas.h>
extern "C" {
    #include <cu_runtime.h>
}

extern "C" error_t *cu_malloc(void **pp, size_t size)
{
    CHECK_NULL_ARGUMENT(pp, "pp");

    hipError_t error = hipMallocManaged(pp, size);
    if (error != hipSuccess)
    {
        return ERROR(ERROR_MEMORY_ALLOCATION,
                     string_create("failed to allocate %zu bytes, %s.", size, hipGetErrorString(error)),
                     NULL);
    }

    return NULL;
}

extern "C" void cu_free(void *p)
{
    hipFree(p);
}

extern "C" error_t *cu_copy(const void *src, void *dst, size_t size)
{
    CHECK_NULL_ARGUMENT(src, "src");
    CHECK_NULL_ARGUMENT(dst, "dst");

    hipMemcpy(dst, src, size, hipMemcpyDeviceToDevice);

    return NULL;
}

extern "C" error_t *cu_addition(datatype_t datatype, uint32_t size, const void *x_data, const void *y_data, void *z_data)
{
    CHECK_NULL_ARGUMENT(x_data, "x_data");
    CHECK_NULL_ARGUMENT(y_data, "y_data");
    CHECK_NULL_ARGUMENT(z_data, "z_data");

    switch (datatype)
    {
    case FLOAT32:
        hipblasScopy(size, (float32_t *) y_data, 1, (float32_t *) z_data, 1); 
        hipblasSaxpy(size, 1.0, (float32_t *) x_data, 1, (float32_t *) z_data, 1);
        break;
    case FLOAT64:
        hipblasDcopy(size, (float64_t *) y_data, 1, (float64_t *) z_data, 1);
        hipblasDaxpy(size, 1.0, (float64_t *) x_data, 1, (float64_t *) z_data, 1);
        break;
    default:
        return ERROR(ERROR_DATATYPE, 
                     string_create("Unsupported datatype %s", datatype_string(datatype)),
                     NULL);    
    }

    return NULL;
}
