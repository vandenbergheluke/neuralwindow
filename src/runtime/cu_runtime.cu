#include <hip/hip_runtime.h>
extern "C" {
    #include <cu_runtime.h>
}

extern "C" error_t *cu_malloc(void **pp, size_t size)
{
    CHECK_NULL(pp, "pp");

    hipError_t error = hipMallocManaged(pp, size);
    if (error != hipSuccess)
        return ERROR(ERROR_MEMORY_ALLOCATION, create_string("failed to allocate %zu bytes, %s.", size, hipGetErrorString(error)), NULL);

    return NULL;
}

extern "C" error_t *cu_free(void *p)
{
    CHECK_NULL(p, "p");

    hipError_t error = hipFree(p);
    if (error != hipSuccess)
        return ERROR(ERROR_MEMORY_FREE, create_string("failed to free memory, %s.", hipGetErrorString(error)), NULL);

    return NULL;
}