#include <hip/hip_runtime.h>
#include <hipblas.h>
extern "C" {
    #include <cu_runtime.h>
}

static hipblasHandle_t handle = NULL;

extern "C" error_t *cu_create_context(void)
{
    hipblasStatus_t status = hipblasCreate(&handle);
    if (status != HIPBLAS_STATUS_SUCCESS)
    {
        return ERROR(ERROR_CREATE,
                     string_create("failed to create cuda context."),
                     NULL);
    }

    return NULL;
}

extern "C" void cu_destroy_context(void)
{
    // TODO: This can return an error but handling it makes tear downs awkward.
    // Most of the sample codes tend to ignore the error as well.
    // We should atleast print the error.
    hipblasDestroy(handle);
}

extern "C" error_t *cu_memory_allocate(void **pp, size_t size)
{
    CHECK_NULL_ARGUMENT(pp, "pp");

    hipError_t error = hipMallocManaged(pp, size);
    if (error != hipSuccess)
    {
        return ERROR(ERROR_MEMORY_ALLOCATION,
                     string_create("failed to allocate %zu bytes %s.", size, hipGetErrorString(error)),
                     NULL);
    }

    return NULL;
}

extern "C" void cu_memory_free(void *p)
{
    // TODO: This can return an error but handling it makes tear downs awkward.
    // Most of the sample codes tend to ignore the error as well.
    // We should atleast print the error.
    hipFree(p);
}

extern "C" error_t *cu_addition(datatype_t datatype, uint32_t size, const void *x_data, const void *y_data, void *z_data)
{
    CHECK_NULL_ARGUMENT(x_data, "x_data");
    CHECK_NULL_ARGUMENT(y_data, "y_data");
    CHECK_NULL_ARGUMENT(z_data, "z_data");

    // TODO: The copy is annoying. Is there a way we can avoid this?
    float32_t alpha_32 = 1.0;
    float64_t alpha_64 = 1.0;
    switch (datatype)
    {
    case FLOAT32:
        hipblasScopy(handle, size, (float32_t *) y_data, 1, (float32_t *) z_data, 1); 
        hipDeviceSynchronize();
        hipblasSaxpy(handle, size, &alpha_32, (float32_t *) x_data, 1, (float32_t *) z_data, 1);
        hipDeviceSynchronize();
        break;
    case FLOAT64:
        hipblasDcopy(handle, size, (float64_t *) y_data, 1, (float64_t *) z_data, 1);
        hipDeviceSynchronize();
        hipblasDaxpy(handle, size, &alpha_64, (float64_t *) x_data, 1, (float64_t *) z_data, 1);
        hipDeviceSynchronize();
        break;
    default:
        return ERROR(ERROR_DATATYPE, 
                     string_create("unsupported datatype %s.", datatype_string(datatype)),
                     NULL);    
    }

    return NULL;
}

extern "C" error_t *cu_matrix_multiplication(datatype_t datatype,
                                             uint32_t m,
                                             uint32_t k,
                                             uint32_t n, 
                                             const void *x_data,
                                             const void *y_data,
                                             void *z_data)
{
    CHECK_NULL_ARGUMENT(x_data, "x_data");
    CHECK_NULL_ARGUMENT(y_data, "y_data");
    CHECK_NULL_ARGUMENT(z_data, "z_data");

    // Note: cuBLAS only accepts column major format hence, to get the
    // matrix product in row major we multiply the matrices that are in
    // row major in reverse order. (AB)^T = C^T = B^TA^T but B and A are 
    // already transposed if they are in row major.
    float32_t beta_32 = 0.0;
    float32_t alpha_32 = 1.0;
    float64_t beta_64 = 0.0;
    float64_t alpha_64 = 1.0;
    switch (datatype)
    {
    case FLOAT32:
        hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, 
                       n, m, k, &alpha_32, (float32_t *) y_data, 
                       n, (float32_t *) x_data, k, &beta_32, (float32_t *) z_data, n);
        hipDeviceSynchronize();
        break;
    case FLOAT64:
        hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, 
                       n, m, k, &alpha_64, (float64_t *) y_data, 
                       n, (float64_t *) x_data, k, &beta_64, (float64_t *) z_data, n);
        hipDeviceSynchronize();
        break;
    default:
        return ERROR(ERROR_DATATYPE, 
                     string_create("unsupported datatype %s.", datatype_string(datatype)),
                     NULL);    
    }

    return NULL;
}