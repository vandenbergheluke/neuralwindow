#include "hip/hip_runtime.h"
/**@file cu_runtime.cu
 * @brief Implementation of low level matrix operations using CUDA kernels and
 * CuBLAS.
 */

#include "magma_types.h"
#include "magmablas_s.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hipsparse.h>
extern "C" {
    #include <cu_runtime.h>
}
#include <magma_v2.h>

#ifndef SYNCHRONOUS
#define SYNCHRONOUS 1
#endif

#define EPSILON 1e-7

#define NW_WARP_SIZE 32

#define NW_NUM_STREAMS 16

#define NW_CHUNK_SIZE 64

// CUDA defns.
static hipblasHandle_t cublas_handle = NULL;
static hipsparseHandle_t cusparse_handle = NULL;
static hipStream_t hip_stream[NW_NUM_STREAMS] = {0};

// MAGMA defns.
static magma_queue_t m_queue[NW_NUM_STREAMS] = {0};

extern "C" nw_error_t *cu_create_context(void)
{
    hipblasStatus_t hipblasStatus_t = hipblasCreate(&cublas_handle);
    if (hipblasStatus_t != HIPBLAS_STATUS_SUCCESS)
    {
        return ERROR(ERROR_CREATE, string_create("failed to create cublas context."), NULL);
    }

    hipsparseStatus_t cusparseStatus = hipsparseCreate(&cusparse_handle);
    if (cusparseStatus != HIPSPARSE_STATUS_SUCCESS)
    {
        return ERROR(ERROR_CREATE, string_create("failed to create cusparse context."), NULL);
    }

    for (int i = 0; i < NW_NUM_STREAMS; ++i)
    {
        hipStreamCreate(&hip_stream[i]);
    }

    magma_int_t error = magma_init();
    if (error != MAGMA_SUCCESS) {
        return ERROR(ERROR_MEMORY_ALLOCATION, string_create("failed to initialize MAGMA.", magma_strerror(error)), NULL);
    }

    magma_device_t m_device;
    magma_getdevice(&m_device);

    for (int i = 0; i < NW_NUM_STREAMS; ++i)
    {
        magma_queue_create_from_cuda(m_device, hip_stream[i], cublas_handle, cusparse_handle, &m_queue[i]);
    }

    return NULL;
}

extern "C" void cu_destroy_context(void)
{
    // Automatically synchronizes the device.
    hipblasDestroy(cublas_handle);
    hipsparseDestroy(cusparse_handle);
    for (int i = 0; i < NW_NUM_STREAMS; ++i)
    {
        hipStreamDestroy(hip_stream[i]);

        magma_queue_destroy(m_queue[i]);
    }
    magma_finalize();
}

extern "C" nw_error_t *cu_memory_allocate(void **pp, size_t size)
{
    CHECK_NULL_ARGUMENT(pp, "pp");

    hipError_t error = hipMallocManaged(pp, size);
    if (error != hipSuccess)
    {
        return ERROR(ERROR_MEMORY_ALLOCATION, string_create("failed to allocate %zu bytes %s.", size, hipGetErrorString(error)), NULL);
    }

    return NULL;
}

extern "C" void cu_memory_free(void *p)
{
    hipFree(p);
}

extern "C" void cu_synchronize(void)
{
    hipDeviceSynchronize();
}

__global__ static void cu_exponential_float32(int n, const float32_t *x_data, int x_stride, float32_t *y_data, int y_stride)
{
    int i = (blockDim.x * blockIdx.x) + threadIdx.x;
    if (i < n)
    {
        y_data[i * y_stride] = expf(x_data[i * x_stride]);
    }
}

__global__ static void cu_exponential_float64(int n, const float64_t *x_data, int x_stride, float64_t *y_data, int y_stride)
{
    int i = (blockDim.x * blockIdx.x) + threadIdx.x;
    if (i < n)
    {
        y_data[i * y_stride] = exp(x_data[i * x_stride]);
    }
}

extern "C" void cu_exponential(datatype_t datatype, int64_t n, const void *x_data, int64_t x_stride, int64_t x_offset, void *y_data, int64_t y_stride, int64_t y_offset)
{
    // CUDA devs want us using ints here for minor optimization purposes, and
    // presumably because we know we're not going to overflow.
    int block_size;
    int grid_size;

    switch (datatype)
    {
    case FLOAT32:
        block_size = NW_WARP_SIZE * 24;

        grid_size = (MIN(NW_CHUNK_SIZE, n) + block_size - 1) / block_size;

        for (int i = 0, j = 0; i < n; i += NW_CHUNK_SIZE, ++j)
        {
            cu_exponential_float32<<<grid_size,
                    block_size,
                    0,
                    hip_stream[j % NW_NUM_STREAMS]>>>(
                        MIN(NW_CHUNK_SIZE, (int) n - i),
                        &((float32_t *) x_data)[x_offset + (i * x_stride)],
                        (int) x_stride,
                        &((float32_t *) y_data)[y_offset + (i * y_stride)],
                        (int) y_stride);
        }

#if SYNCHRONOUS
        hipDeviceSynchronize();
#endif
        break;
    case FLOAT64:
        block_size = NW_WARP_SIZE * 24;

        grid_size = (MIN(NW_CHUNK_SIZE, n) + block_size - 1) / block_size;

        for (int i = 0, j = 0; i < n; i += NW_CHUNK_SIZE, ++j)
        {
            cu_exponential_float64<<<grid_size,
                    block_size,
                    0,
                    hip_stream[j % NW_NUM_STREAMS]>>>(
                        MIN(NW_CHUNK_SIZE, (int) n - i),
                        &((float64_t *) x_data)[x_offset + (i * x_stride)],
                        (int) x_stride,
                        &((float64_t *) y_data)[y_offset + (i * y_stride)],
                        (int) y_stride);
        }

#if SYNCHRONOUS
        hipDeviceSynchronize();
#endif
        break;
    default:
        break;
    }
}

__global__ static void cu_logarithm_float32(int n, const float32_t *x_data, int x_stride, float32_t *y_data, int y_stride)
{
    int i = (blockDim.x * blockIdx.x) + threadIdx.x;
    if (i < n)
    {
        y_data[i * y_stride] = logf(x_data[i * x_stride]);
    }
}

__global__ static void cu_logarithm_float64(int n, const float64_t *x_data, int x_stride, float64_t *y_data, int y_stride)
{
    int i = (blockDim.x * blockIdx.x) + threadIdx.x;
    if (i < n)
    {
        y_data[i * y_stride] = log(x_data[i * x_stride]);
    }
}

extern "C" void cu_logarithm(datatype_t datatype, int64_t n, const void *x_data, int64_t x_stride, int64_t x_offset, void *y_data, int64_t y_stride, int64_t y_offset)
{
    int block_size;
    int grid_size;

    switch (datatype)
    {
    case FLOAT32:
        block_size = NW_WARP_SIZE * 24;

        grid_size = (MIN(NW_CHUNK_SIZE, n) + block_size - 1) / block_size;

        for (int i = 0, j = 0; i < n; i += NW_CHUNK_SIZE, ++j)
        {
            cu_logarithm_float32<<<grid_size,
                    block_size,
                    0,
                    hip_stream[j % NW_NUM_STREAMS]>>>(
                        MIN(NW_CHUNK_SIZE, (int) n - i),
                        &((float32_t *) x_data)[x_offset + (i * x_stride)],
                        (int) x_stride,
                        &((float32_t *) y_data)[y_offset + (i * y_stride)],
                        (int) y_stride);
        }

#if SYNCHRONOUS
        hipDeviceSynchronize();
#endif
        break;

    case FLOAT64:
        block_size = NW_WARP_SIZE * 24;

        grid_size = (MIN(NW_CHUNK_SIZE, n) + block_size - 1) / block_size;

        for (int i = 0, j = 0; i < n; i += NW_CHUNK_SIZE, ++j)
        {
            cu_logarithm_float64<<<grid_size,
                    block_size,
                    0,
                    hip_stream[j % NW_NUM_STREAMS]>>>(
                        MIN(NW_CHUNK_SIZE, (int) n - i),
                        &((float64_t *) x_data)[x_offset + (i * x_stride)],
                        (int) x_stride,
                        &((float64_t *) y_data)[y_offset + (i * y_stride)],
                        (int) y_stride);
        }

#if SYNCHRONOUS
        hipDeviceSynchronize();
#endif
        break;
    default:
        break;
    }
}

__global__ static void cu_sine_float32(int n, const float32_t *x_data, int x_stride, float32_t *y_data, int y_stride)
{
    int i = (blockDim.x * blockIdx.x) + threadIdx.x;
    if (i < n)
    {
        y_data[i * y_stride] = sinf(x_data[i * x_stride]);
    }
}

__global__ static void cu_sine_float64(int n, const float64_t *x_data, int x_stride, float64_t *y_data, int y_stride)
{
    int i = (blockDim.x * blockIdx.x) + threadIdx.x;
    if (i < n)
    {
        y_data[i * y_stride] = sin(x_data[i * x_stride]);
    }
}

extern "C" void cu_sine(datatype_t datatype, int64_t n, const void *x_data, int64_t x_stride, int64_t x_offset, void *y_data, int64_t y_stride, int64_t y_offset)
{
    int block_size;
    int grid_size;

    switch (datatype)
    {
    case FLOAT32:
        block_size = NW_WARP_SIZE * 24;

        grid_size = (MIN(NW_CHUNK_SIZE, n) + block_size - 1) / block_size;

        for (int i = 0, j = 0; i < n; i += NW_CHUNK_SIZE, ++j)
        {
            cu_sine_float32<<<grid_size,
                    block_size,
                    0,
                    hip_stream[j % NW_NUM_STREAMS]>>>(
                        MIN(NW_CHUNK_SIZE, (int) n - i),
                        &((float32_t *) x_data)[x_offset + (i * x_stride)],
                        (int) x_stride,
                        &((float32_t *) y_data)[y_offset + (i * y_stride)],
                        (int) y_stride);
        }

#if SYNCHRONOUS
        hipDeviceSynchronize();
#endif
        break;
    case FLOAT64:
        block_size = NW_WARP_SIZE * 24;

        grid_size = (MIN(NW_CHUNK_SIZE, n) + block_size - 1) / block_size;

        for (int i = 0, j = 0; i < n; i += NW_CHUNK_SIZE, ++j)
        {
            cu_sine_float64<<<grid_size,
                    block_size,
                    0,
                    hip_stream[j % NW_NUM_STREAMS]>>>(
                        MIN(NW_CHUNK_SIZE, (int) n - i),
                        &((float64_t *) x_data)[x_offset + (i * x_stride)],
                        (int) x_stride,
                        &((float64_t *) y_data)[y_offset + (i * y_stride)],
                        (int) y_stride);
        }

#if SYNCHRONOUS
        hipDeviceSynchronize();
#endif
        break;
    default:
        break;
    }
}

__global__ static void cu_cosine_float32(int n, const float32_t *x_data, int x_stride, float32_t *y_data, int y_stride)
{
    int i = (blockDim.x * blockIdx.x) + threadIdx.x;
    if (i < n)
    {
        y_data[i * y_stride] = cosf(x_data[i * x_stride]);
    }
}

__global__ static void cu_cosine_float64(int n, const float64_t *x_data, int x_stride, float64_t *y_data, int y_stride)
{
    int i = (blockDim.x * blockIdx.x) + threadIdx.x;
    if (i < n)
    {
        y_data[i * y_stride] = cos(x_data[i * x_stride]);
    }
}

extern "C" void cu_cosine(datatype_t datatype, int64_t n, const void *x_data, int64_t x_stride, int64_t x_offset, void *y_data, int64_t y_stride, int64_t y_offset)
{
    int block_size;
    int grid_size;

    switch (datatype)
    {
    case FLOAT32:
        block_size = NW_WARP_SIZE * 24;

        grid_size = (MIN(NW_CHUNK_SIZE, n) + block_size - 1) / block_size;

        for (int i = 0, j = 0; i < n; i += NW_CHUNK_SIZE, ++j)
        {
            cu_cosine_float32<<<grid_size,
                    block_size,
                    0,
                    hip_stream[j % NW_NUM_STREAMS]>>>(
                        MIN(NW_CHUNK_SIZE, (int) n - i),
                        &((float32_t *) x_data)[x_offset + (i * x_stride)],
                        (int) x_stride,
                        &((float32_t *) y_data)[y_offset + (i * y_stride)],
                        (int) y_stride);
        }

#if SYNCHRONOUS
        hipDeviceSynchronize();
#endif
        break;
    case FLOAT64:
        block_size = NW_WARP_SIZE * 24;

        grid_size = (MIN(NW_CHUNK_SIZE, n) + block_size - 1) / block_size;

        for (int i = 0, j = 0; i < n; i += NW_CHUNK_SIZE, ++j)
        {
            cu_cosine_float64<<<grid_size,
                    block_size,
                    0,
                    hip_stream[j % NW_NUM_STREAMS]>>>(
                        MIN(NW_CHUNK_SIZE, (int) n - i),
                        &((float64_t *) x_data)[x_offset + (i * x_stride)],
                        (int) x_stride,
                        &((float64_t *) y_data)[y_offset + (i * y_stride)],
                        (int) y_stride);
        }

#if SYNCHRONOUS
        hipDeviceSynchronize();
#endif
        break;
    default:
        break;
    }
}

__global__ static void cu_square_root_float32(int n, const float32_t *x_data, int x_stride, float32_t *y_data, int y_stride)
{
    int i = (blockDim.x * blockIdx.x) + threadIdx.x;
    if (i < n)
    {
        y_data[i * y_stride] = sqrtf(x_data[i * x_stride]);
    }
}

__global__ static void cu_square_root_float64(int n, const float64_t *x_data, int x_stride, float64_t *y_data, int y_stride)
{
    int i = (blockDim.x * blockIdx.x) + threadIdx.x;
    if (i < n)
    {
        y_data[i * y_stride] = sqrt(x_data[i * x_stride]);
    }
}

extern "C" void cu_square_root(datatype_t datatype, int64_t n, const void *x_data, int64_t x_stride, int64_t x_offset, void *y_data, int64_t y_stride, int64_t y_offset)
{
    int block_size;
    int grid_size;

    switch (datatype)
    {
    case FLOAT32:
        block_size = NW_WARP_SIZE * 24;

        grid_size = (MIN(NW_CHUNK_SIZE, n) + block_size - 1) / block_size;

        for (int i = 0, j = 0; i < n; i += NW_CHUNK_SIZE, ++j)
        {
            cu_square_root_float32<<<grid_size,
                    block_size,
                    0,
                    hip_stream[j % NW_NUM_STREAMS]>>>(
                        MIN(NW_CHUNK_SIZE, (int) n - i),
                        &((float32_t *) x_data)[x_offset + (i * x_stride)],
                        (int) x_stride,
                        &((float32_t *) y_data)[y_offset + (i * y_stride)],
                        (int) y_stride);
        }

#if SYNCHRONOUS
        hipDeviceSynchronize();
#endif
        break;
    case FLOAT64:
        block_size = NW_WARP_SIZE * 24;

        grid_size = (MIN(NW_CHUNK_SIZE, n) + block_size - 1) / block_size;

        for (int i = 0, j = 0; i < n; i += NW_CHUNK_SIZE, ++j)
        {
            cu_square_root_float64<<<grid_size,
                    block_size,
                    0,
                    hip_stream[j % NW_NUM_STREAMS]>>>(
                        MIN(NW_CHUNK_SIZE, (int) n - i),
                        &((float64_t *) x_data)[x_offset + (i * x_stride)],
                        (int) x_stride,
                        &((float64_t *) y_data)[y_offset + (i * y_stride)],
                        (int) y_stride);
        }

#if SYNCHRONOUS
        hipDeviceSynchronize();
#endif
        break;
    default:
        break;
    }
}

__global__ static void cu_reciprocal_float32(int n, const float32_t *x_data, int x_stride, float32_t *y_data, int y_stride)
{
    int i = (blockDim.x * blockIdx.x) + threadIdx.x;
    if (i < n)
    {
        y_data[i * y_stride] = 1. / x_data[i * x_stride];
    }
}

__global__ static void cu_reciprocal_float64(int n, const float64_t *x_data, int x_stride, float64_t *y_data, int y_stride)
{
    int i = (blockDim.x * blockIdx.x) + threadIdx.x;
    if (i < n)
    {
        y_data[i * y_stride] = 1. / x_data[i * x_stride];
    }
}

extern "C" void cu_reciprocal(datatype_t datatype, int64_t n, const void *x_data, int64_t x_stride, int64_t x_offset, void *y_data, int64_t y_stride, int64_t y_offset)
{
    int block_size;
    int grid_size;

    switch (datatype)
    {
    case FLOAT32:
        block_size = NW_WARP_SIZE * 24;

        grid_size = (MIN(NW_CHUNK_SIZE, n) + block_size - 1) / block_size;

        for (int i = 0, j = 0; i < n; i += NW_CHUNK_SIZE, ++j)
        {
            cu_reciprocal_float32<<<grid_size,
                    block_size,
                    0,
                    hip_stream[j % NW_NUM_STREAMS]>>>(
                        MIN(NW_CHUNK_SIZE, (int) n - i),
                        &((float32_t *) x_data)[x_offset + (i * x_stride)],
                        (int) x_stride,
                        &((float32_t *) y_data)[y_offset + (i * y_stride)],
                        (int) y_stride);
        }

#if SYNCHRONOUS
        hipDeviceSynchronize();
#endif
        break;
    case FLOAT64:
        block_size = NW_WARP_SIZE * 24;

        grid_size = (MIN(NW_CHUNK_SIZE, n) + block_size - 1) / block_size;

        for (int i = 0, j = 0; i < n; i += NW_CHUNK_SIZE, ++j)
        {
            cu_reciprocal_float64<<<grid_size,
                    block_size,
                    0,
                    hip_stream[j % NW_NUM_STREAMS]>>>(
                        MIN(NW_CHUNK_SIZE, (int) n - i),
                        &((float64_t *) x_data)[x_offset + (i * x_stride)],
                        (int) x_stride,
                        &((float64_t *) y_data)[y_offset + (i * y_stride)],
                        (int) y_stride);
        }

#if SYNCHRONOUS
        hipDeviceSynchronize();
#endif
        break;
    default:
        break;
    }
}

extern "C" void cu_copy(datatype_t datatype, int64_t n, const void *x_data, int64_t x_stride, int64_t x_offset, void *y_data, int64_t y_stride, int64_t y_offset)
{
    // TODO: We don't have any way to measure the performance of these functions
    // yet. We'll use MAGMA for now.
    switch (datatype)
    {
    case FLOAT32:
#if 1
        hipDeviceSynchronize();
        magma_scopy((magma_int_t) n, &((magmaFloat_const_ptr) x_data)[x_offset], (magma_int_t) x_stride, &((magmaFloat_ptr) y_data)[y_offset], (magma_int_t) y_stride, m_queue[0]);
        magma_queue_sync(m_queue[0]);
#else
        hipblasScopy(cublas_handle, (int) n, &((float32_t *) x_data)[x_offset], (int) x_stride, &((float32_t *) y_data)[y_offset], (int) y_stride);
        hipDeviceSynchronize();
#endif
        break;
    case FLOAT64:
#if 1
        hipDeviceSynchronize();
        magma_dcopy((magma_int_t) n, &((magmaDouble_const_ptr) x_data)[x_offset], (magma_int_t) x_stride, &((magmaDouble_ptr) y_data)[y_offset], (magma_int_t) y_stride, m_queue[0]);
        magma_queue_sync(m_queue[0]);
#else
        hipblasDcopy(cublas_handle, (int) n, &((float64_t *) x_data)[x_offset], (int) x_stride, &((float64_t *) y_data)[y_offset], (int) y_stride);
        hipDeviceSynchronize();
#endif
        break;
    default:
        break;
    }
}

__global__ static void cu_negation_float32(int n, const float32_t *x_data, int x_stride, float32_t *y_data, int y_stride)
{
    int i = (blockDim.x * blockIdx.x) + threadIdx.x;
    if (i < n)
    {
        y_data[i * y_stride] = -x_data[i * x_stride];
    }
}

__global__ static void cu_negation_float64(int n, const float64_t *x_data, int x_stride, float64_t *y_data, int y_stride)
{
    int i = (blockDim.x * blockIdx.x) + threadIdx.x;
    if (i < n)
    {
        y_data[i * y_stride] = -x_data[i * x_stride];
    }
}

void cu_negation(datatype_t datatype,
                 int64_t n,
                 const void *x_data,
                 int64_t x_stride,
                 int64_t x_offset,
                 void *y_data,
                 int64_t y_stride,
                 int64_t y_offset)
{
    int block_size;
    int grid_size;

    switch (datatype)
    {
    case FLOAT32:
        block_size = NW_WARP_SIZE * 24;

        grid_size = (MIN(NW_CHUNK_SIZE, n) + block_size - 1) / block_size;

        for (int i = 0, j = 0; i < n; i += NW_CHUNK_SIZE, ++j)
        {
            cu_negation_float32<<<grid_size,
                    block_size,
                    0,
                    hip_stream[j % NW_NUM_STREAMS]>>>(
                        MIN(NW_CHUNK_SIZE, (int) n - i),
                        &((float32_t *) x_data)[x_offset + (i * x_stride)],
                        (int) x_stride,
                        &((float32_t *) y_data)[y_offset + (i * y_stride)],
                        (int) y_stride);
        }

#if SYNCHRONOUS
        hipDeviceSynchronize();
#endif
        break;
    case FLOAT64:
        block_size = NW_WARP_SIZE * 24;

        grid_size = (MIN(NW_CHUNK_SIZE, n) + block_size - 1) / block_size;

        for (int i = 0, j = 0; i < n; i += NW_CHUNK_SIZE, ++j)
        {
            cu_negation_float64<<<grid_size,
                    block_size,
                    0,
                    hip_stream[j % NW_NUM_STREAMS]>>>(
                        MIN(NW_CHUNK_SIZE, (int) n - i),
                        &((float64_t *) x_data)[x_offset + (i * x_stride)],
                        (int) x_stride,
                        &((float64_t *) y_data)[y_offset + (i * y_stride)],
                        (int) y_stride);
        }

#if SYNCHRONOUS
        hipDeviceSynchronize();
#endif
        break;
    default:
        break;
    }
}

__global__ static void cu_rectified_linear_float32(int n, const float32_t *x_data, int x_stride, float32_t *y_data, int y_stride)
{
    int i = (blockDim.x * blockIdx.x) + threadIdx.x;
    if (i < n)
    {
        float32_t value = x_data[i * x_stride];
        y_data[i * y_stride] = (value > 0.0) ? value : (float32_t) 0.0;
    }
}

__global__ static void cu_rectified_linear_float64(int n, const float64_t *x_data, int x_stride, float64_t *y_data, int y_stride)
{
    int i = (blockDim.x * blockIdx.x) + threadIdx.x;
    if (i < n)
    {
        float64_t value = x_data[i * x_stride];
        y_data[i * y_stride] = (value > 0.0) ? value : (float64_t) 0.0;
    }
}

extern "C" void cu_rectified_linear(datatype_t datatype, int64_t n, const void *x_data, int64_t x_stride, int64_t x_offset, void *y_data, int64_t y_stride, int64_t y_offset)
{
    int block_size;
    int grid_size;

    switch (datatype)
    {
    case FLOAT32:
        block_size = NW_WARP_SIZE * 24;

        grid_size = (MIN(NW_CHUNK_SIZE, n) + block_size - 1) / block_size;

        for (int i = 0, j = 0; i < n; i += NW_CHUNK_SIZE, ++j)
        {
            cu_rectified_linear_float32<<<grid_size,
                    block_size,
                    0,
                    hip_stream[j % NW_NUM_STREAMS]>>>(
                        MIN(NW_CHUNK_SIZE, (int) n - i),
                        &((float32_t *) x_data)[x_offset + (i * x_stride)],
                        (int) x_stride,
                        &((float32_t *) y_data)[y_offset + (i * y_stride)],
                        (int) y_stride);
        }

#if SYNCHRONOUS
        hipDeviceSynchronize();
#endif
        break;
    case FLOAT64:
        block_size = NW_WARP_SIZE * 24;

        grid_size = (MIN(NW_CHUNK_SIZE, n) + block_size - 1) / block_size;

        for (int i = 0, j = 0; i < n; i += NW_CHUNK_SIZE, ++j)
        {
            cu_rectified_linear_float64<<<grid_size,
                    block_size,
                    0,
                    hip_stream[j % NW_NUM_STREAMS]>>>(
                        MIN(NW_CHUNK_SIZE, (int) n - i),
                        &((float64_t *) x_data)[x_offset + (i * x_stride)],
                        (int) x_stride,
                        &((float64_t *) y_data)[y_offset + (i * y_stride)],
                        (int) y_stride);
        }

#if SYNCHRONOUS
        hipDeviceSynchronize();
#endif
        break;
    default:
        break;
    }
}

__global__ static void cu_sigmoid_float32(int n, const float32_t *x_data, int x_stride, float32_t *y_data, int y_stride)
{
    int i = (blockDim.x * blockIdx.x) + threadIdx.x;
    if (i < n)
    {
        float32_t x = x_data[i * x_stride];
        y_data[i * y_stride] = (float32_t) 1.0 / ((float32_t) 1.0 + expf(-x));
    }
}

__global__ static void cu_sigmoid_float64(int n, const float64_t *x_data, int x_stride, float64_t *y_data, int y_stride)
{
    int i = (blockDim.x * blockIdx.x) + threadIdx.x;
    if (i < n)
    {
        float64_t x = x_data[i * x_stride];
        y_data[i * y_stride] = (float64_t) 1.0 / ((float64_t) 1.0 + exp(-x));
    }
}

extern "C" void cu_sigmoid(datatype_t datatype, int64_t n, const void *x_data, int64_t x_stride, int64_t x_offset, void *y_data, int64_t y_stride, int64_t y_offset)
{
    int block_size;
    int grid_size;

    switch (datatype)
    {
    case FLOAT32:
        block_size = NW_WARP_SIZE * 24;

        grid_size = (MIN(NW_CHUNK_SIZE, n) + block_size - 1) / block_size;

        for (int i = 0, j = 0; i < n; i += NW_CHUNK_SIZE, ++j)
        {
            cu_sigmoid_float32<<<grid_size,
                    block_size,
                    0,
                    hip_stream[j % NW_NUM_STREAMS]>>>(
                        MIN(NW_CHUNK_SIZE, (int) n - i),
                        &((float32_t *) x_data)[x_offset + (i * x_stride)],
                        (int) x_stride,
                        &((float32_t *) y_data)[y_offset + (i * y_stride)],
                        (int) y_stride);
        }

#if SYNCHRONOUS
        hipDeviceSynchronize();
#endif
        break;
    case FLOAT64:
        block_size = NW_WARP_SIZE * 24;

        grid_size = (MIN(NW_CHUNK_SIZE, n) + block_size - 1) / block_size;

        for (int i = 0, j = 0; i < n; i += NW_CHUNK_SIZE, ++j)
        {
            cu_sigmoid_float64<<<grid_size,
                    block_size,
                    0,
                    hip_stream[j % NW_NUM_STREAMS]>>>(
                        MIN(NW_CHUNK_SIZE, (int) n - i),
                        &((float64_t *) x_data)[x_offset + (i * x_stride)],
                        (int) x_stride,
                        &((float64_t *) y_data)[y_offset + (i * y_stride)],
                        (int) y_stride);
        }

#if SYNCHRONOUS
        hipDeviceSynchronize();
#endif
        break;
    default:
        break;
    }
}

extern "C" static void cu_addition_float32(int n,
                                           const float32_t *x_data,
                                           int x_stride,
                                           const float32_t *y_data,
                                           int y_stride,
                                           float32_t *z_data,
                                           int z_stride)
{
    float alpha = 1.0;
    hipDeviceSynchronize();
    magma_scopy((magma_int_t) n, (magmaFloat_const_ptr) x_data, (magma_int_t) x_stride, (magmaFloat_ptr) z_data, (magma_int_t) z_stride, m_queue[0]);
    magma_queue_sync(m_queue[0]);
    magma_saxpy((magma_int_t) n, alpha, (magmaFloat_const_ptr) y_data, (magma_int_t) y_stride, (magmaFloat_ptr) z_data, (magma_int_t) z_stride, m_queue[0]);
    magma_queue_sync(m_queue[0]);
}

extern "C" static void cu_addition_float64(int n,
                                           const float64_t *x_data,
                                           int x_stride,
                                           const float64_t *y_data,
                                           int y_stride,
                                           double *z_data,
                                           float64_t z_stride)
{
    double alpha = 1.0;
    hipDeviceSynchronize();
    magma_dcopy((magma_int_t) n, (magmaDouble_const_ptr) x_data, (magma_int_t) x_stride, (magmaDouble_ptr) z_data, (magma_int_t) z_stride, m_queue[0]);
    magma_queue_sync(m_queue[0]);
    magma_daxpy((magma_int_t) n, alpha, (magmaDouble_const_ptr) y_data, (magma_int_t) y_stride, (magmaDouble_ptr) z_data, (magma_int_t) z_stride, m_queue[0]);
    magma_queue_sync(m_queue[0]);
}

extern "C" void cu_addition(datatype_t datatype,
                            int64_t n,
                            const void *x_data,
                            int64_t x_stride,
                            int64_t x_offset,
                            const void *y_data,
                            int64_t y_stride,
                            int64_t y_offset,
                            void *z_data,
                            int64_t z_stride,
                            int64_t z_offset)
{
    switch (datatype)
    {
    case FLOAT32:
        cu_addition_float32((int) n, 
                            &((float32_t *) x_data)[x_offset], 
                            (int) x_stride,
                            &((float32_t *) y_data)[y_offset],
                            (int) y_stride,
                            &((float32_t *) z_data)[z_offset],
                            (int) z_stride);
        break;
    case FLOAT64:
        cu_addition_float64((int) n, 
                            &((float64_t *) x_data)[x_offset], 
                            (int) x_stride,
                            &((float64_t *) y_data)[y_offset],
                            (int) y_stride,
                            &((float64_t *) z_data)[z_offset],
                            (int) z_stride);
        break;
    default:
        break;
    }
}

extern "C" static void cu_subtraction_float32(int n,
                                              const float32_t *x_data,
                                              int x_stride,
                                              const float32_t *y_data,
                                              int y_stride,
                                              float32_t *z_data,
                                              int z_stride)
{
    float alpha = -1.0;
    hipDeviceSynchronize();
    magma_scopy((magma_int_t) n, (magmaFloat_const_ptr) x_data, (magma_int_t) x_stride, (magmaFloat_ptr) z_data, (magma_int_t) z_stride, m_queue[0]);
    magma_queue_sync(m_queue[0]);
    magma_saxpy((magma_int_t) n, alpha, (magmaFloat_const_ptr) y_data, (magma_int_t) y_stride, (magmaFloat_ptr) z_data, (magma_int_t) z_stride, m_queue[0]);
    magma_queue_sync(m_queue[0]);
}

extern "C" static void cu_subtraction_float64(int n,
                                              const float64_t *x_data,
                                              int x_stride,
                                              const float64_t *y_data,
                                              int y_stride,
                                              float64_t *z_data,
                                              int z_stride)
{
    double alpha = -1.0;
    hipDeviceSynchronize();
    magma_dcopy((magma_int_t) n, (magmaDouble_const_ptr) x_data, (magma_int_t) x_stride, (magmaDouble_ptr) z_data, (magma_int_t) z_stride, m_queue[0]);
    magma_queue_sync(m_queue[0]);
    magma_daxpy((magma_int_t) n, alpha, (magmaDouble_const_ptr) y_data, (magma_int_t) y_stride, (magmaDouble_ptr) z_data, (magma_int_t) z_stride, m_queue[0]);
    magma_queue_sync(m_queue[0]);
}

extern "C" void cu_subtraction(datatype_t datatype,
                               int64_t n,
                               const void *x_data,
                               int64_t x_stride,
                               int64_t x_offset,
                               const void *y_data,
                               int64_t y_stride,
                               int64_t y_offset,
                               void *z_data,
                               int64_t z_stride,
                               int64_t z_offset)
{
    switch (datatype)
    {
    case FLOAT32:
        cu_subtraction_float32((int) n, 
                               &((float32_t *) x_data)[x_offset], 
                               (int) x_stride,
                               &((float32_t *) y_data)[y_offset],
                               (int) y_stride,
                               &((float32_t *) z_data)[z_offset],
                               (int) z_stride);
        break;
    case FLOAT64:
        cu_subtraction_float64((int) n, 
                               &((float64_t *) x_data)[x_offset], 
                               (int) x_stride,
                               &((float64_t *) y_data)[y_offset],
                               (int) y_stride,
                               &((float64_t *) z_data)[z_offset],
                               (int) z_stride);
        break;
    default:
        break;
    }
}

__global__ static void cu_multiplication_float32(int n,
                                                 const float32_t *x_data,
                                                 int x_stride,
                                                 const float32_t *y_data,
                                                 int y_stride,
                                                 float32_t *z_data,
                                                 int z_stride)
{
    int i = (blockDim.x * blockIdx.x) + threadIdx.x;
    if (i < n)
    {
        z_data[i * z_stride] = x_data[i * x_stride] * y_data[i * y_stride];
    }
}

__global__ static void cu_multiplication_float64(int n,
                                                 const float64_t *x_data,
                                                 int x_stride,
                                                 const float64_t *y_data,
                                                 int y_stride,
                                                 float64_t *z_data,
                                                 int z_stride)
{
    int i = (blockDim.x * blockIdx.x) + threadIdx.x;
    if (i < n)
    {
        z_data[i * z_stride] = x_data[i * x_stride] * y_data[i * y_stride];
    }
}

extern "C" void cu_multiplication(datatype_t datatype,
                                  int64_t n,
                                  const void *x_data,
                                  int64_t x_stride,
                                  int64_t x_offset,
                                  const void *y_data,
                                  int64_t y_stride,
                                  int64_t y_offset,
                                  void *z_data,
                                  int64_t z_stride,
                                  int64_t z_offset)
{
    int block_size;
    int grid_size;

    switch (datatype)
    {
    case FLOAT32:
        block_size = NW_WARP_SIZE * 24;

        grid_size = (MIN(NW_CHUNK_SIZE, n) + block_size - 1) / block_size;

        for (int i = 0, j = 0; i < n; i += NW_CHUNK_SIZE, ++j)
        {
            cu_multiplication_float32<<<grid_size,
                    block_size,
                    0,
                    hip_stream[j % NW_NUM_STREAMS]>>>(
                        MIN(NW_CHUNK_SIZE, (int) n - i),
                        &((float32_t *) x_data)[x_offset + (i * x_stride)],
                        (int) x_stride,
                        &((float32_t *) y_data)[y_offset + (i * y_stride)],
                        (int) y_stride,
                        &((float32_t *) z_data)[z_offset + (i * z_stride)],
                        (int) z_stride);
        }

#if SYNCHRONOUS
        hipDeviceSynchronize();
#endif
        break;
    case FLOAT64:
        block_size = NW_WARP_SIZE * 24;

        grid_size = (MIN(NW_CHUNK_SIZE, n) + block_size - 1) / block_size;

        for (int i = 0, j = 0; i < n; i += NW_CHUNK_SIZE, ++j)
        {
            cu_multiplication_float64<<<grid_size,
                    block_size,
                    0,
                    hip_stream[j % NW_NUM_STREAMS]>>>(
                        MIN(NW_CHUNK_SIZE, (int) n - i),
                        &((float64_t *) x_data)[x_offset + (i * x_stride)],
                        (int) x_stride,
                        &((float64_t *) y_data)[y_offset + (i * y_stride)],
                        (int) y_stride,
                        &((float64_t *) z_data)[z_offset + (i * z_stride)],
                        (int) z_stride);
        }

#if SYNCHRONOUS
        hipDeviceSynchronize();
#endif
        break;
    default:
        break;
    }
}

__global__ static void cu_division_float32(int n, const float32_t *x_data, int x_stride, const float32_t *y_data, int y_stride, float32_t *z_data, int z_stride)
{
    int i = (blockDim.x * blockIdx.x) + threadIdx.x;
    if (i < n)
    {
        z_data[i * z_stride] = x_data[i * x_stride] / y_data[i * y_stride];
    }
}

__global__ static void cu_division_float64(int n, const float64_t *x_data, int x_stride, const float64_t *y_data, int y_stride, float64_t *z_data, int z_stride)
{
    int i = (blockDim.x * blockIdx.x) + threadIdx.x;
    if (i < n)
    {
        z_data[i * z_stride] = x_data[i * x_stride] / y_data[i * y_stride];
    }
}

extern "C" void cu_division(datatype_t datatype, int64_t n, const void *x_data, int64_t x_stride, int64_t x_offset, const void *y_data, int64_t y_stride, int64_t y_offset, void *z_data, int64_t z_stride, int64_t z_offset)
{
    int block_size;
    int grid_size;

    switch (datatype)
    {
    case FLOAT32:
        block_size = NW_WARP_SIZE * 24;

        grid_size = (MIN(NW_CHUNK_SIZE, n) + block_size - 1) / block_size;

        for (int i = 0, j = 0; i < n; i += NW_CHUNK_SIZE, ++j)
        {
            cu_division_float32<<<grid_size,
                    block_size,
                    0,
                    hip_stream[j % NW_NUM_STREAMS]>>>(
                        MIN(NW_CHUNK_SIZE, (int) n - i),
                        &((float32_t *) x_data)[x_offset + (i * x_stride)],
                        (int) x_stride,
                        &((float32_t *) y_data)[y_offset + (i * y_stride)],
                        (int) y_stride,
                        &((float32_t *) z_data)[z_offset + (i * z_stride)],
                        (int) z_stride);
        }

#if SYNCHRONOUS
        hipDeviceSynchronize();
#endif
        break;
    case FLOAT64:
        block_size = NW_WARP_SIZE * 24;

        grid_size = (MIN(NW_CHUNK_SIZE, n) + block_size - 1) / block_size;

        for (int i = 0, j = 0; i < n; i += NW_CHUNK_SIZE, ++j)
        {
            cu_division_float64<<<grid_size,
                    block_size,
                    0,
                    hip_stream[j % NW_NUM_STREAMS]>>>(
                        MIN(NW_CHUNK_SIZE, (int) n - i),
                        &((float64_t *) x_data)[x_offset + (i * x_stride)],
                        (int) x_stride,
                        &((float64_t *) y_data)[y_offset + (i * y_stride)],
                        (int) y_stride,
                        &((float64_t *) z_data)[z_offset + (i * z_stride)],
                        (int) z_stride);
        }

#if SYNCHRONOUS
        hipDeviceSynchronize();
#endif
        break;
    default:
        break;
    }
}

__global__ static void cu_power_float32(int n, const float32_t *x_data, int x_stride, const float32_t *y_data, int y_stride, float32_t *z_data, int z_stride)
{
    int i = (blockDim.x * blockIdx.x) + threadIdx.x;
    if (i < n)
    {
        z_data[i * z_stride] = powf(x_data[i * x_stride], y_data[i * y_stride]);
    }
}

__global__ static void cu_power_float64(int n, const float64_t *x_data, int x_stride, const float64_t *y_data, int y_stride, float64_t *z_data, int z_stride)
{
    int i = (blockDim.x * blockIdx.x) + threadIdx.x;
    if (i < n)
    {
        z_data[i * z_stride] = pow(x_data[i * x_stride], y_data[i * y_stride]);
    }
}

extern "C" void cu_power(datatype_t datatype, int64_t n, const void *x_data, int64_t x_stride, int64_t x_offset, const void *y_data, int64_t y_stride, int64_t y_offset, void *z_data, int64_t z_stride, int64_t z_offset)
{
    int block_size;
    int grid_size;

    switch (datatype)
    {
    case FLOAT32:
        block_size = NW_WARP_SIZE * 24;

        grid_size = (MIN(NW_CHUNK_SIZE, n) + block_size - 1) / block_size;

        for (int i = 0, j = 0; i < n; i += NW_CHUNK_SIZE, ++j)
        {
            cu_power_float32<<<grid_size,
                    block_size,
                    0,
                    hip_stream[j % NW_NUM_STREAMS]>>>(
                        MIN(NW_CHUNK_SIZE, (int) n - i),
                        &((float32_t *) x_data)[x_offset + (i * x_stride)],
                        (int) x_stride,
                        &((float32_t *) y_data)[y_offset + (i * y_stride)],
                        (int) y_stride,
                        &((float32_t *) z_data)[z_offset + (i * z_stride)],
                        (int) z_stride);
        }

#if SYNCHRONOUS
        hipDeviceSynchronize();
#endif
        break;
    case FLOAT64:
        block_size = NW_WARP_SIZE * 24;

        grid_size = (MIN(NW_CHUNK_SIZE, n) + block_size - 1) / block_size;

        for (int i = 0, j = 0; i < n; i += NW_CHUNK_SIZE, ++j)
        {
            cu_power_float64<<<grid_size,
                    block_size,
                    0,
                    hip_stream[j % NW_NUM_STREAMS]>>>(
                        MIN(NW_CHUNK_SIZE, (int) n - i),
                        &((float64_t *) x_data)[x_offset + (i * x_stride)],
                        (int) x_stride,
                        &((float64_t *) y_data)[y_offset + (i * y_stride)],
                        (int) y_stride,
                        &((float64_t *) z_data)[z_offset + (i * z_stride)],
                        (int) z_stride);
        }

#if SYNCHRONOUS
        hipDeviceSynchronize();
#endif
        break;
    default:
        break;
    }
}

__global__ static void cu_compare_equal_float32(int n, const float32_t *x_data, int x_stride, const float32_t *y_data, int y_stride, float32_t *z_data, int z_stride)
{
    float32_t x, y;
    int i = (blockDim.x * blockIdx.x) + threadIdx.x;
    if (i < n)
    {
        x = x_data[i * x_stride];
        y = y_data[i * y_stride];
        z_data[i * z_stride] = fabsf(x - y) < EPSILON ? (float32_t) 1.0 : (float32_t) 0.0;
    }
}

__global__ static void cu_compare_equal_float64(int n, const float64_t *x_data, int x_stride, const float64_t *y_data, int y_stride, float64_t *z_data, int z_stride)
{
    float64_t x, y;
    int i = (blockDim.x * blockIdx.x) + threadIdx.x;
    if (i < n)
    {
        x = x_data[i * x_stride];
        y = y_data[i * y_stride];
        z_data[i * z_stride] = fabs(x - y) < EPSILON ? (float64_t) 1.0 : (float64_t) 0.0;
    }
}

extern "C" void cu_compare_equal(datatype_t datatype, int64_t n, const void *x_data, int64_t x_stride, int64_t x_offset, const void *y_data, int64_t y_stride, int64_t y_offset, void *z_data, int64_t z_stride, int64_t z_offset)
{
    int block_size;
    int grid_size;

    switch (datatype)
    {
    case FLOAT32:
        block_size = NW_WARP_SIZE * 24;

        grid_size = (MIN(NW_CHUNK_SIZE, n) + block_size - 1) / block_size;

        for (int i = 0, j = 0; i < n; i += NW_CHUNK_SIZE, ++j)
        {
            cu_compare_equal_float32<<<grid_size,
                    block_size,
                    0,
                    hip_stream[j % NW_NUM_STREAMS]>>>(
                        MIN(NW_CHUNK_SIZE, (int) n - i),
                        &((float32_t *) x_data)[x_offset + (i * x_stride)],
                        (int) x_stride,
                        &((float32_t *) y_data)[y_offset + (i * y_stride)],
                        (int) y_stride,
                        &((float32_t *) z_data)[z_offset + (i * z_stride)],
                        (int) z_stride);
        }

#if SYNCHRONOUS
        hipDeviceSynchronize();
#endif
        break;
    case FLOAT64:
        block_size = NW_WARP_SIZE * 24;

        grid_size = (MIN(NW_CHUNK_SIZE, n) + block_size - 1) / block_size;

        for (int i = 0, j = 0; i < n; i += NW_CHUNK_SIZE, ++j)
        {
            cu_compare_equal_float64<<<grid_size,
                    block_size,
                    0,
                    hip_stream[j % NW_NUM_STREAMS]>>>(
                        MIN(NW_CHUNK_SIZE, (int) n - i),
                        &((float64_t *) x_data)[x_offset + (i * x_stride)],
                        (int) x_stride,
                        &((float64_t *) y_data)[y_offset + (i * y_stride)],
                        (int) y_stride,
                        &((float64_t *) z_data)[z_offset + (i * z_stride)],
                        (int) z_stride);
        }

#if SYNCHRONOUS
        hipDeviceSynchronize();
#endif
        break;
    default:
        break;
    }
}

__global__ static void cu_compare_greater_float32(int n, const float32_t *x_data, int x_stride, const float32_t *y_data, int y_stride, float32_t *z_data, int z_stride)
{
    int i = (blockDim.x * blockIdx.x) + threadIdx.x;
    if (i < n)
    {
        z_data[i * z_stride] = (x_data[i * x_stride] > y_data[i * y_stride]) ? (float32_t) 1.0 : (float32_t) 0.0;
    }
}

__global__ static void cu_compare_greater_float64(int n, const float64_t *x_data, int x_stride, const float64_t *y_data, int y_stride, float64_t *z_data, int z_stride)
{
    int i = (blockDim.x * blockIdx.x) + threadIdx.x;
    if (i < n)
    {
        z_data[i * z_stride] = (x_data[i * x_stride] > y_data[i * y_stride]) ? (float64_t) 1.0 : (float64_t) 0.0;
    }
}

extern "C" void cu_compare_greater(datatype_t datatype, int64_t n, const void *x_data, int64_t x_stride, int64_t x_offset, const void *y_data, int64_t y_stride, int64_t y_offset, void *z_data, int64_t z_stride, int64_t z_offset)
{
    int block_size;
    int grid_size;

    switch (datatype)
    {
    case FLOAT32:
        block_size = NW_WARP_SIZE * 24;

        grid_size = (MIN(NW_CHUNK_SIZE, n) + block_size - 1) / block_size;

        for (int i = 0, j = 0; i < n; i += NW_CHUNK_SIZE, ++j)
        {
            cu_compare_greater_float32<<<grid_size,
                    block_size,
                    0,
                    hip_stream[j % NW_NUM_STREAMS]>>>(
                        MIN(NW_CHUNK_SIZE, (int) n - i),
                        &((float32_t *) x_data)[x_offset + (i * x_stride)],
                        (int) x_stride,
                        &((float32_t *) y_data)[y_offset + (i * y_stride)],
                        (int) y_stride,
                        &((float32_t *) z_data)[z_offset + (i * z_stride)],
                        (int) z_stride);
        }

#if SYNCHRONOUS
        hipDeviceSynchronize();
#endif
        break;
    case FLOAT64:
        block_size = NW_WARP_SIZE * 24;

        grid_size = (MIN(NW_CHUNK_SIZE, n) + block_size - 1) / block_size;

        for (int i = 0, j = 0; i < n; i += NW_CHUNK_SIZE, ++j)
        {
            cu_compare_greater_float64<<<grid_size,
                    block_size,
                    0,
                    hip_stream[j % NW_NUM_STREAMS]>>>(
                        MIN(NW_CHUNK_SIZE, (int) n - i),
                        &((float64_t *) x_data)[x_offset + (i * x_stride)],
                        (int) x_stride,
                        &((float64_t *) y_data)[y_offset + (i * y_stride)],
                        (int) y_stride,
                        &((float64_t *) z_data)[z_offset + (i * z_stride)],
                        (int) z_stride);
        }

#if SYNCHRONOUS
        hipDeviceSynchronize();
#endif
        break;
    default:
        break;
    }
}

extern "C" void cu_matrix_multiplication_float32(datatype_t datatype,
                                                 int64_t m,
                                                 int64_t k,
                                                 int64_t n,
                                                 bool_t x_transpose,
                                                 bool_t y_transpose,
                                                 const float32_t *x_data,
                                                 const float32_t *y_data,
                                                 float32_t *z_data)
{
    float alpha = 1.0;
    float beta = 0.0;
    hipDeviceSynchronize();
    magma_sgemm(x_transpose ? MagmaTrans : MagmaNoTrans,
            y_transpose ? MagmaTrans : MagmaNoTrans,
            n, m, k, alpha, (magmaFloat_const_ptr) y_data,
            n, (magmaFloat_const_ptr) x_data, k, beta,
            (magmaFloat_ptr) z_data, n, m_queue[0]);
    magma_queue_sync(m_queue[0]);
}

extern "C" void cu_matrix_multiplication_float64(datatype_t datatype,
                                                 int64_t m,
                                                 int64_t k,
                                                 int64_t n,
                                                 bool_t x_transpose,
                                                 bool_t y_transpose,
                                                 const float64_t *x_data,
                                                 const float64_t *y_data,
                                                 float64_t *z_data)
{
    double alpha = 1.0;
    double beta = 0.0;
    hipDeviceSynchronize();
    magma_dgemm(x_transpose ? MagmaTrans : MagmaNoTrans,
            y_transpose ? MagmaTrans : MagmaNoTrans,
            n, m, k, alpha, (magmaDouble_const_ptr) y_data,
            n, (magmaDouble_const_ptr) x_data, k, beta,
            (magmaDouble_ptr) z_data, n, m_queue[0]);
    magma_queue_sync(m_queue[0]);
}

extern "C" void cu_matrix_multiplication(datatype_t datatype,
                                         int64_t m,
                                         int64_t k,
                                         int64_t n,
                                         bool_t x_transpose,
                                         bool_t y_transpose,
                                         const void *x_data,
                                         int64_t x_offset,
                                         const void *y_data,
                                         int64_t y_offset,
                                         void *z_data,
                                         int64_t z_offset)
{
    switch (datatype)
    {
    case FLOAT32:
        cu_matrix_multiplication_float32(datatype,
                                         m,
                                         k,
                                         n,
                                         x_transpose,
                                         y_transpose,
                                         &((float32_t *) x_data)[x_offset],
                                         &((float32_t *) y_data)[y_offset],
                                         &((float32_t *) z_data)[z_offset]);
        break;
    case FLOAT64:
        cu_matrix_multiplication_float64(datatype,
                                         m,
                                         k,
                                         n,
                                         x_transpose,
                                         y_transpose,
                                         &((float64_t *) x_data)[x_offset],
                                         &((float64_t *) y_data)[y_offset],
                                         &((float64_t *) z_data)[z_offset]);
        break;
    default:
        break;
    }
}

extern "C" static void cu_summation_float32(int n, const float32_t *x_data, int x_stride, float32_t *y_data)
{
    // This one is a tossup with cublas in terms of performance, and we have a
    // bit of a blindspot when it comes to smaller matrices, but we'll use
    // MAGMA for now.
    float32_t *temp;
    hipMallocManaged((void **) &temp, sizeof(float32_t));
    *temp = (float32_t) 1.0;
    hipDeviceSynchronize();
    hipblasSdot(cublas_handle, n, x_data, x_stride, temp, 0, y_data);
    hipDeviceSynchronize();
    hipFree(temp);
}

extern "C" static void cu_summation_float64(int n, const float64_t *x_data, int x_stride, float64_t *y_data)
{
    // This one is a tossup with cublas in terms of performance, and we have a
    // bit of a blindspot when it comes to smaller matrices, but we'll use
    // MAGMA for now.
    float64_t *temp;
    hipMallocManaged((void **) &temp, sizeof(float64_t));
    *temp = (float64_t) 1.0;
    hipDeviceSynchronize();
    hipblasDdot(cublas_handle, n, x_data, x_stride, temp, 0, y_data);
    hipDeviceSynchronize();
    hipFree(temp);
}

extern "C" void cu_summation(datatype_t datatype, int64_t n, const void *x_data, int64_t x_stride, int64_t x_offset, void *y_data, int64_t y_offset)
{
    switch (datatype)
    {
    case FLOAT32:
        cu_summation_float32((int) n, &((float32_t *) x_data)[x_offset], (int) x_stride, &((float32_t *) y_data)[y_offset]);
        break;
    case FLOAT64:
        cu_summation_float64((int) n, &((float64_t *) x_data)[x_offset], (int) x_stride, &((float64_t *) y_data)[y_offset]);
        break;
    default:
        break;
    }
}

// Since atomicMAX doesn't normally support float
__device__ static float32_t atomicMAX(float32_t *address, float32_t val)
{
    int* address_as_i = (int*) address;
    int old = *address_as_i;
    int assumed;
    do {
        assumed = old;
        old = atomicCAS(address_as_i, assumed,
            __float_as_int(fmaxf(val, __int_as_float(assumed))));
    } while (assumed != old);
    return __int_as_float(old);
}

__device__ static float64_t atomicMAX(float64_t *address, float64_t val)
{
    long long unsigned* address_as_ull = (long long unsigned*) address;
    long long unsigned old = *address_as_ull;
    long long unsigned assumed;
    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed,
            __double_as_longlong(fmax(val, __longlong_as_double(assumed))));
    } while (assumed != old);
    return __longlong_as_double(old);
}

__global__ static void cu_maximum_float32(int n, const float32_t *x_data, int x_stride, float32_t *y_data)
{
    __shared__ float32_t current_maximum;
    current_maximum = *x_data;
    __syncthreads();
    int i = (blockDim.x * blockIdx.x) + threadIdx.x + 1;
    if (i < n)
    {
        atomicMAX(&current_maximum, x_data[i * x_stride]);
    }
    __syncthreads();
    *y_data = current_maximum;
}

__global__ static void cu_maximum_float64(int n, const float64_t *x_data, int x_stride, float64_t *y_data)
{
    __shared__ float64_t current_maximum;
    current_maximum = *x_data;
    __syncthreads();
    int i = (blockDim.x * blockIdx.x) + threadIdx.x + 1;
    if (i < n)
    {
        atomicMAX(&current_maximum, x_data[i * x_stride]);
    }
    __syncthreads();
    *y_data = current_maximum;
}

extern "C" void cu_maximum(datatype_t datatype, int64_t n, const void *x_data, int64_t x_stride, int64_t x_offset, void *y_data, int64_t y_offset)
{
    int block_size;
    int grid_size;

    switch (datatype)
    {
    case FLOAT32:
        block_size = NW_WARP_SIZE * 24;

        grid_size = (MIN(NW_CHUNK_SIZE, n) + block_size - 1) / block_size;

        // TODO: Figure out a good way to dispatch in chunks
        hipDeviceSynchronize();
        cu_maximum_float32<<<grid_size, block_size>>>((int) n, &((float32_t *) x_data)[x_offset], (int) x_stride, &((float32_t *) y_data)[y_offset]);

#if SYNCHRONOUS
        hipDeviceSynchronize();
#endif
        break;
    case FLOAT64:
        block_size = NW_WARP_SIZE * 24;

        grid_size = (MIN(NW_CHUNK_SIZE, n) + block_size - 1) / block_size;

        // TODO: Figure out a good way to dispatch in chunks
        hipDeviceSynchronize();
        cu_maximum_float64<<<grid_size, block_size>>>((int) n, &((float64_t *) x_data)[x_offset], (int) x_stride, &((float64_t *) y_data)[y_offset]);

#if SYNCHRONOUS
        hipDeviceSynchronize();
#endif
        break;
    default:
        break;
    }
}
