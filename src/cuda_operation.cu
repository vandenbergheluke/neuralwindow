#include <hip/hip_runtime.h>
extern "C" {
    #include <cuda_operation.h>
}

extern "C" 
error_t cuda_malloc(buffer_t *buffer_ptr, size_t size)
{
    if (buffer_ptr == NULL)
    {
        return STATUS_NULL_POINTER;
    }
    hipError_t error = hipMalloc((void **) buffer_ptr, size);
    if (error != hipSuccess)
    {
        printf("error:%s:%s:%d:%d:%s\n", __FILE__, __FUNCTION__, __LINE__, error, hipGetErrorString(error));
        return STATUS_MEMORY_ALLOCATION_FAILURE;
    }
    return STATUS_SUCCESS;
}

extern "C"
error_t cuda_free(buffer_t buffer)
{
    hipError_t error = hipFree((void *) buffer);
    if (error != hipSuccess)
    {
        printf("error:%s:%s:%d:%d:%s\n", __FILE__, __FUNCTION__, __LINE__, error, hipGetErrorString(error));
        return STATUS_MEMORY_FREE_FAILURE;
    }
    return STATUS_SUCCESS;
}